#include "hip/hip_runtime.h"
﻿/*****************************************************************//**
 * \file   cuda_matrix.cu
 * \brief  CUDA-accelerated matrix operations implementation providing
		   optimized matrix computations and manipulations on GPU.
 * \author AndrewElizabeth (easterghost1@gmail.com)
 * \date   February 2025
 *********************************************************************/
#include "cuda_matrix.cuh"

template <class T>
static int autoSetBlockSize(T func)
{
	int blockSize = 0;
	int gridSize = 0;
	hipOccupancyMaxPotentialBlockSize(&gridSize, &blockSize, func);
	//if (blockSize == 0)
	//	throw runtime_error("Failed to set block size.");
	return max(blockSize, 32);
}

template <class T>
static dim3 autoSetBlockSize2D(T func, const int rows, const int cols)
{
	int blockSize = 0;
	int gridSize = 0;
	hipOccupancyMaxPotentialBlockSize(&gridSize, &blockSize, func);
	if (blockSize == 0)
		throw runtime_error("Failed to set block size.");
	return dim3(blockSize, 1);
}

template <typename Type>
cumatrix<Type>::cumatrix() : rows(0), cols(0), mat(nullptr) {}

template <typename Type>
cumatrix<Type>::cumatrix(const uint32_t rows, const uint32_t cols) : rows(rows), cols(cols)
{
	allocate_matrix(rows, cols);
	hipMemset(mat, 0, static_cast<size_type>(rows) * cols * sizeof(value_type));
	hipblasCreate(&handle);
	hipsolverDnCreate(&solver_handle);
}

template <typename Type>
cumatrix<Type>::cumatrix(const uint32_t rows, const uint32_t cols, const MatrixType type) : rows(rows), cols(cols)
{
	clock_t start = clock();
	size_t total_elements = static_cast<size_t>(rows) * cols;
	hipMalloc((void**)&mat, total_elements * sizeof(value_type));
	int blockSize = 0;
	int gridSize = 0;
	hiprandStatePhilox4_32_10_t* states = nullptr;
	hipDeviceSynchronize();
	time_used_init += clock() - start;
	switch (type)
	{
	case Zero:
		hipMemset(mat, 0, total_elements * sizeof(value_type));
		break;
	case Ones:
		blockSize = autoSetBlockSize(ones_matrix_kernel<value_type>);
		gridSize = static_cast<int>((total_elements + blockSize - 1) / (size_t)blockSize);
		ones_matrix_kernel<value_type> << <gridSize, blockSize >> > (mat, total_elements);
		hipDeviceSynchronize();
		break;
	case Identity:
		if (rows != cols)
			throw runtime_error("Identity matrix must be square matrix.");
		hipMemset(mat, 0, static_cast<size_t>(rows) * cols * sizeof(value_type));
		blockSize = autoSetBlockSize(identity_matrix_kernel<value_type>);
		gridSize = (rows + blockSize - 1) / blockSize;
		identity_matrix_kernel<value_type> << <gridSize, blockSize >> > (mat, rows);
		hipDeviceSynchronize();
		break;
	case Random:
		start = clock();
		blockSize = autoSetBlockSize(setup_random_kernel);
		//cout << "Block size of setup random kernel: " << blockSize << endl;
		gridSize = static_cast<int>((total_elements + blockSize - 1) / (size_t)blockSize);
		hipMalloc((void**)&states, total_elements * sizeof(hiprandStatePhilox4_32_10_t));
		setup_random_kernel << <gridSize, blockSize >> > (states, time(0), total_elements);
		hipDeviceSynchronize();
		time_used_gen_init += clock() - start;
		start = clock();
		if constexpr (is_same<value_type, float>::value)
		{
			time_used_switch_type += clock() - start;
			start = clock();
			blockSize = autoSetBlockSize(float_random_matrix_kernel);
			//cout << "Block size of float random kernel: " << blockSize << endl;
			gridSize = static_cast<int>((total_elements + blockSize - 1) / (size_t)blockSize);
			time_used_setblock += clock() - start;
			start = clock();
			float_random_matrix_kernel << <gridSize, blockSize >> >
				((float*)mat, total_elements, states);
			hipDeviceSynchronize();
			time_used_gen += clock() - start;
			start = clock();
		}
		else if constexpr (is_same<value_type, double>::value)
		{
			time_used_switch_type += clock() - start;
			start = clock();
			blockSize = autoSetBlockSize(double_random_matrix_kernel);
			//cout << "Block size of double random kernel: " << blockSize << endl;
			gridSize = (total_elements + blockSize - 1) / blockSize;
			time_used_setblock += clock() - start;
			start = clock();
			double_random_matrix_kernel << <gridSize, blockSize >> >
				((double*)mat, total_elements, states);
			hipDeviceSynchronize();
			time_used_gen += clock() - start;
			start = clock();
		}
		else if constexpr (is_same<value_type, int>::value)
		{
			time_used_switch_type += clock() - start;
			start = clock();
			blockSize = autoSetBlockSize(int_random_matrix_kernel);
			//cout << "Block size of int random kernel: " << blockSize << endl;
			gridSize = static_cast<int>((total_elements + blockSize - 1) / (size_t)blockSize);
			time_used_setblock += clock() - start;
			start = clock();
			int_random_matrix_kernel << <gridSize, blockSize >> >
				((int*)mat, total_elements, states);
			hipDeviceSynchronize();
			time_used_gen += clock() - start;
			start = clock();
		}
		else
		{
			total_elements = static_cast<size_t>(rows) * cols * sizeof(value_type) / sizeof(int);
			//hipFree(states);
			hipMalloc((void**)&states, total_elements * sizeof(hiprandStatePhilox4_32_10_t));
			setup_random_kernel << <gridSize, blockSize >> > (states, time(0), total_elements);
			hipDeviceSynchronize();
			time_used_gen_init += clock() - start;
			start = clock();
			blockSize = autoSetBlockSize(int_random_matrix_kernel);
			//cout << "Block size of int random kernel: " << blockSize << endl;
			gridSize = (total_elements + blockSize - 1) / blockSize;
			time_used_setblock += clock() - start;
			start = clock();
			int_random_matrix_kernel << <gridSize, blockSize >> >
				((int*)mat, total_elements, states);
			hipDeviceSynchronize();
			time_used_gen += clock() - start;
			start = clock();
		}
		hipFree(states);
		break;
		/*
	case QuasiRandom:
		start = clock();
		if constexpr (is_same<value_type, float>::value)
		{
			blockSize = autoSetBlockSize(setup_q32random_kernel);
			//cout << "Block size of setup random kernel: " << blockSize << endl;
			gridSize = (total_elements + blockSize - 1) / blockSize;
			hipMalloc((void**)&qstates32, total_elements * sizeof(hiprandStateScrambledSobol32_t));
			hipMalloc((void**)&dr_vec32, cols * sizeof(hiprandDirectionVectors32_t));
			hiprandGetDirectionVectors32(&dr_vec32, HIPRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6);
			setup_q32random_kernel << <gridSize, blockSize >> > (qstates32, dr_vec32, rows, cols);
			hipDeviceSynchronize();
			time_used_gen_init += clock() - start;
			start = clock();
			blockSize = autoSetBlockSize(float_qrandom_matrix_kernel);
			//cout << "Block size of float random kernel: " << blockSize << endl;
			gridSize = (total_elements + blockSize - 1) / blockSize;
			time_used_setblock += clock() - start;
			start = clock();
			float_qrandom_matrix_kernel << <gridSize, blockSize >> > ((float*)mat, qstates32, rows, cols);
			hipDeviceSynchronize();
			time_used_gen += clock() - start;
			start = clock();
			hipFree(qstates32);
			hipFree(dr_vec32);
		}
		else if constexpr (is_same<value_type, double>::value)
		{
			blockSize = autoSetBlockSize(setup_q64random_kernel);
			//cout << "Block size of setup random kernel: " << blockSize << endl;
			gridSize = (total_elements + blockSize - 1) / blockSize;
			hipMalloc((void**)&qstates64, total_elements * sizeof(hiprandStateScrambledSobol64_t));
			hipMalloc((void**)&dr_vec64, cols * sizeof(hiprandDirectionVectors64_t));
			hiprandGetDirectionVectors64(&dr_vec64, HIPRAND_SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6);
			setup_q64random_kernel << <gridSize, blockSize >> > (qstates64, dr_vec64, rows, cols);
			hipDeviceSynchronize();
			time_used_gen_init += clock() - start;
			start = clock();
			blockSize = autoSetBlockSize(double_qrandom_matrix_kernel);
			//cout << "Block size of double random kernel: " << blockSize << endl;
			gridSize = (total_elements + blockSize - 1) / blockSize;
			time_used_setblock += clock() - start;
			start = clock();
			double_qrandom_matrix_kernel << <gridSize, blockSize >> >
				((double*)mat, qstates64, rows, cols);
			hipDeviceSynchronize();
			time_used_gen += clock() - start;
			start = clock();
			hipFree(qstates64);
			hipFree(dr_vec64);
		}
		else if constexpr (is_same<value_type, int>::value)
		{
			blockSize = autoSetBlockSize(setup_q32random_kernel);
			//cout << "Block size of setup random kernel: " << blockSize << endl;
			//system("pause");
			gridSize = (total_elements + blockSize - 1) / blockSize;
			hipMalloc((void**)&qstates32, total_elements * sizeof(hiprandStateScrambledSobol32_t));
			hipMalloc((void**)&dr_vec32, cols * sizeof(hiprandDirectionVectors32_t));
			hiprandGetDirectionVectors32(&dr_vec32, HIPRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6);
			setup_q32random_kernel << <gridSize, blockSize >> > (qstates32, dr_vec32, rows, cols);
			hipDeviceSynchronize();
			time_used_gen_init += clock() - start;
			start = clock();
			blockSize = autoSetBlockSize(int_qrandom_matrix_kernel);
			cout << "Block size of int random kernel: " << blockSize << endl;
			gridSize = (total_elements + blockSize - 1) / blockSize;
			time_used_setblock += clock() - start;
			start = clock();
			int_qrandom_matrix_kernel << <gridSize, blockSize >> > ((int*)mat, qstates32, rows, cols);
			hipDeviceSynchronize();
			time_used_gen += clock() - start;
			start = clock();
			hipFree(qstates32);
			hipFree(dr_vec32);
		}
		else
		{
			total_elements = static_cast<size_t>(rows) * cols * sizeof(value_type) / sizeof(int);
			hipMalloc((void**)&qstates32, total_elements * sizeof(hiprandStateScrambledSobol32_t));
			hipMalloc((void**)&dr_vec32, cols * sizeof(hiprandDirectionVectors32_t));
			hiprandGetDirectionVectors32(&dr_vec32, HIPRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6);
			setup_q32random_kernel << <gridSize, blockSize >> > (qstates32, dr_vec32, rows, cols);
			hipDeviceSynchronize();
			time_used_gen_init += clock() - start;
			start = clock();
			blockSize = autoSetBlockSize(int_qrandom_matrix_kernel);
			//cout << "Block size of int random kernel: " << blockSize << endl;
			gridSize = (total_elements + blockSize - 1) / blockSize;
			time_used_setblock += clock() - start;
			start = clock();
			int_qrandom_matrix_kernel << <gridSize, blockSize >> >
				((int*)mat, qstates32, rows, cols);
			hipDeviceSynchronize();
			time_used_gen += clock() - start;
			start = clock();
			hipFree(qstates32);
			hipFree(dr_vec32);
		}
		break;
		*/
	default:
		throw runtime_error("Unknown matrix type.");
	}
	hipDeviceSynchronize();
	hipblasCreate(&handle);
	hipsolverDnCreate(&solver_handle);
	time_used_end += clock() - start;
}

template <typename Type>
cumatrix<Type>::cumatrix(const uint32_t size) : cumatrix(size, size) {}

template <typename Type>
cumatrix<Type>::cumatrix(const uint32_t size, const MatrixType type) : cumatrix(size, size, type) {}

template <typename Type>
cumatrix<Type>::cumatrix(const uint32_t rows, const uint32_t cols, const pointer src) : cumatrix(rows, cols) { hipMemcpy(mat, src, static_cast<size_t>(rows) * cols * sizeof(value_type), hipMemcpyHostToDevice); }

template<typename Type>
cumatrix<Type>::cumatrix(const uint32_t rows, const uint32_t cols, const vector<value_type>& src) : cumatrix(rows, cols, src.data()) {}

template <typename Type>
cumatrix<Type>::cumatrix(const uint32_t size, const pointer src) : cumatrix(size, size, src) {}

template<typename Type>
cumatrix<Type>::cumatrix(const uint32_t size, const vector<value_type>& src) : cumatrix(size, size, src.data()) {}

template<typename Type>
cumatrix<Type>::cumatrix(cumatrix&& other) noexcept
{
	rows = other.rows;
	cols = other.cols;
	mat = other.mat;
	other.rows = 0;
	other.cols = 0;
	other.mat = nullptr;
	handle = other.handle;
	solver_handle = other.solver_handle;
	other.handle = nullptr;
	other.solver_handle = nullptr;
}

template <typename Type>
cumatrix<Type>::cumatrix(const cumatrix<value_type>& other) : rows(other.rows), cols(other.cols)
{
	int total_elements = rows * cols;
	hipMalloc((void**)&mat, total_elements * sizeof(value_type));
	hipMemcpy(mat, other.mat, total_elements * sizeof(value_type), hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();
	hipblasCreate(&handle);
	hipsolverDnCreate(&solver_handle);
}

template<typename Type>
cumatrix<Type>& cumatrix<Type>::operator=(const cumatrix<value_type>& other)
{
	if (this == &other)
		return *this;
	if (mat != nullptr)
		hipFree(mat);
	rows = other.rows;
	cols = other.cols;
	hipMalloc((void**)&mat, static_cast<size_t>(rows) * cols * sizeof(value_type));
	hipMemcpy(mat, other.mat, static_cast<size_t>(rows) * cols * sizeof(value_type), hipMemcpyDeviceToDevice);
	hipblasCreate(&handle);
	hipsolverDnCreate(&solver_handle);
	return *this;
}

template<typename Type>
cumatrix<Type>& cumatrix<Type>::operator=(cumatrix&& other) noexcept
{
	if (this == &other)
		return *this;
	if (mat != nullptr)
		hipFree(mat);
	rows = other.rows;
	cols = other.cols;
	mat = other.mat;
	other.rows = 0;
	other.cols = 0;
	other.mat = nullptr;
	handle = other.handle;
	solver_handle = other.solver_handle;
	other.handle = nullptr;
	other.solver_handle = nullptr;
	return *this;
}

template <typename Type>
cumatrix<Type>::~cumatrix()
{
	if (IS_SAFE_DATA)
		hipMemset(mat, 0, static_cast<size_t>(rows) * cols * sizeof(value_type));
	hipFree(mat);
	rows = 0;
	cols = 0;
	mat = nullptr;
	//hipDeviceSynchronize();
	//hipFree(handle);
	//hipFree(solver_handle);
	hipblasDestroy(handle);
	hipsolverDnDestroy(solver_handle);
}

template<typename Type>
bool cumatrix<Type>::operator==(const cumatrix<value_type>& other) const
{
	if (rows != other.rows || cols != other.cols)
		return false;
	int total_elements = rows * cols;
	int blockSize = autoSetBlockSize(elementwise_equal_kernel<value_type>);
	int gridSize = (total_elements + blockSize - 1) / blockSize;
	char* res = nullptr;
	hipMalloc((void**)&res, total_elements * sizeof(char));
	elementwise_equal_kernel<value_type> << <gridSize, blockSize >> >
		(mat, other.mat, res, total_elements);
	hipDeviceSynchronize();
	vector<char> host_res(total_elements);
	hipMemcpy(host_res.data(), res, total_elements * sizeof(char), hipMemcpyDeviceToHost);
	bool result = all_of(host_res.begin(), host_res.end(), [](char x) { return x; });
	hipFree(res);
	return result;
}

template<typename Type>
bool cumatrix<Type>::operator!=(const cumatrix<value_type>& other) const
{
	if (rows != other.rows || cols != other.cols)
		return true;
	int total_elements = rows * cols;
	int blockSize = autoSetBlockSize(elementwise_equal_kernel<value_type>);
	int gridSize = (total_elements + blockSize - 1) / blockSize;
	char* res = nullptr;
	hipMalloc((void**)&res, total_elements * sizeof(char));
	elementwise_equal_kernel<value_type> << <gridSize, blockSize >> >
		(mat, other.mat, res, total_elements);
	hipDeviceSynchronize();
	vector<char> host_res(total_elements);
	hipMemcpy(host_res.data(), res, total_elements * sizeof(bool), hipMemcpyDeviceToHost);
	bool result = any_of(host_res.begin(), host_res.end(), [](char x) { return !x; });
	hipFree(res);
	return result;
}

template<typename Type>
size_t cumatrix<Type>::size() const noexcept { return static_cast<size_t>(rows) * cols; }

template<typename Type>
void cumatrix<Type>::reserve(const size_type new_cap)
{
	if (new_cap <= capacity())
		return;
	pointer tmp = nullptr;
	hipMalloc((void**)&tmp, new_cap * sizeof(value_type));
	hipMemcpy(tmp, mat, static_cast<size_t>(rows) * cols * sizeof(value_type), hipMemcpyDeviceToDevice);
	hipFree(mat);
	mat = tmp;
	tmp = nullptr;
}

template<typename Type>
bool cumatrix<Type>::empty() const noexcept { return (mat == nullptr || size() == 0); }

template<typename Type>
constexpr size_t cumatrix<Type>::max_size() const noexcept { return numeric_limits<size_t>::max() / sizeof(value_type); }

template<typename Type>Type& cumatrix<Type>::front()
{
	if (empty())
		throw runtime_error("Matrix is empty.");
	value_type res = 0;
	hipMemcpy(&res, mat, sizeof(value_type), hipMemcpyDeviceToHost);
	return res;
}

template<typename Type>
const Type& cumatrix<Type>::front() const
{
	if (empty())
		throw runtime_error("Matrix is empty.");
	value_type res = 0;
	hipMemcpy(&res, mat, sizeof(value_type), hipMemcpyDeviceToHost);
	return res;
}

template<typename Type>
Type& cumatrix<Type>::back()
{
	if (empty())
		throw runtime_error("Matrix is empty.");
	value_type res = 0;
	hipMemcpy(&res, mat + static_cast<size_t>(rows) * cols - 1, sizeof(value_type), hipMemcpyDeviceToHost);
	return res;
}

template<typename Type>
const Type& cumatrix<Type>::back() const
{
	if (empty())
		throw runtime_error("Matrix is empty.");
	value_type res = 0;
	hipMemcpy(&res, mat + static_cast<size_t>(rows) * cols - 1, sizeof(value_type), hipMemcpyDeviceToHost);
	return res;
}

template<typename Type>
Type cumatrix<Type>::operator[](const coord_t coord) const
{
	if (coord.x >= rows || coord.y >= cols)
		throw out_of_range("Index out of range.");
	value_type res = 0;
	hipMemcpy(&res, mat + coord.x * rows + coord.y, sizeof(value_type), hipMemcpyDeviceToHost);
	return res;
}

template<typename Type>
Type cumatrix<Type>::operator[](const uint32_t index) const
{
	if (index >= static_cast<size_t>(rows) * cols)
		throw out_of_range("Index out of range.");
	value_type res = 0;
	hipMemcpy(&res, mat + index, sizeof(value_type), hipMemcpyDeviceToHost);
	return res;
}

template<typename Type>
void cumatrix<Type>::clear() noexcept
{
	if (IS_SAFE_DATA) hipMemset(mat, 0, static_cast<size_t>(rows) * cols * sizeof(value_type));
	hipFree(mat);
	rows = 0;
	cols = 0;
	mat = nullptr;
}

template<typename Type>
void cumatrix<Type>::swap(cumatrix<value_type>& other) noexcept
{
	swap(rows, other.rows);
	swap(cols, other.cols);
	swap(mat, other.mat);
}

template<typename Type>
void cumatrix<Type>::assign(const cumatrix<value_type>& other)
{
	if (this == &other)
		return;
	rows = other.rows;
	cols = other.cols;
	if (mat != nullptr)
		hipFree(mat);
	hipMalloc((void**)&mat, static_cast<size_t>(rows) * cols * sizeof(value_type));
	hipMemcpy(mat, other.mat, static_cast<size_t>(rows) * cols * sizeof(value_type), hipMemcpyDeviceToDevice);
}

template<typename Type>
void cumatrix<Type>::assign(cumatrix<value_type>&& other) noexcept
{
	if (this == &other)
		return;
	if (mat != nullptr)
		hipFree(mat);
	rows = other.rows;
	cols = other.cols;
	mat = other.mat;
	other.rows = 0;
	other.cols = 0;
	other.mat = nullptr;
}

template<typename Type>
void cumatrix<Type>::assign(const uint32_t rows, const uint32_t cols, const_reference val)
{
	if (mat != nullptr)
		hipFree(mat);
	this->rows = rows;
	this->cols = cols;
	hipMalloc((void**)&mat, static_cast<size_t>(rows) * cols * sizeof(value_type));
	int blockSize = autoSetBlockSize(assign_kernel<value_type>);
	int gridSize = (rows * cols + blockSize - 1) / blockSize;
	assign_kernel<value_type> << <gridSize, blockSize >> >
		(mat, val, rows * cols);
	hipDeviceSynchronize();
}

template<typename Type>
void cumatrix<Type>::assign(const uint32_t size, const_reference val) { assign(size, size, val); }

template<typename Type>
void cumatrix<Type>::assign(const initializer_list<value_type>& il)
{
	if (mat != nullptr)
		hipFree(mat);
	rows = il.size();
	cols = 1;
	hipMalloc((void**)&mat, static_cast<size_t>(rows) * cols * sizeof(value_type));
	hipMemcpy(mat, il.begin(), static_cast<size_t>(rows) * cols * sizeof(value_type), hipMemcpyHostToDevice);
}

template<typename Type>
void cumatrix<Type>::insert(const uint32_t rows, const uint32_t cols, const_reference val)
{
	if (rows == 0 || cols == 0)
		throw runtime_error("Invalid matrix size.");
	if (this->rows * this->cols < rows * cols)
	{
		resize(rows, cols);
		set(rows, cols, val);
		return;
	}
}

template<typename Type>
Type cumatrix<Type>::at(const uint32_t rows, const uint32_t cols) const
{
	if (rows >= this->rows || cols >= this->cols)
		throw out_of_range("Index out of range.");
	value_type res = 0;
	hipMemcpy(&res, mat + rows * this->cols + cols, sizeof(value_type), hipMemcpyDeviceToHost);
	return res;
}

template<typename Type>
size_t cumatrix<Type>::capacity() const noexcept { return static_cast<size_t>(rows) * cols; }

template<typename Type>
void cumatrix<Type>::set(const uint32_t row, const uint32_t col, const value_type value)
{
	if (row >= rows || col >= cols)
		throw out_of_range("Index out of range.");
	hipMemcpy(mat + row * cols + col, &value, sizeof(value_type), hipMemcpyHostToDevice);
}

template<typename Type>
Type* cumatrix<Type>::data() noexcept { return this->mat; }

template<typename Type>
const Type* cumatrix<Type>::data() const noexcept { return const_cast<pointer>(this->mat); }

template<typename Type>
void cumatrix<Type>::print_matrix()
{
	pointer host_data = new value_type[rows * cols];
	hipMemcpy(host_data, mat, static_cast<size_t>(rows) * cols * sizeof(value_type), hipMemcpyDeviceToHost);
	for (uint32_t i = 0; i < rows; i++)
	{
		for (uint32_t j = 0; j < cols; j++)
			cout << host_data[i * cols + j] << " ";
		cout << endl;
	}
	if (IS_SAFE_DATA) memset(host_data, 0, static_cast<size_t>(rows) * cols * sizeof(value_type));
	delete[] host_data;
}

template<typename Type>
string cumatrix<Type>::to_string() const
{
	pointer host_data = new value_type[rows * cols];
	hipMemcpy(host_data, mat, static_cast<size_t>(rows) * cols * sizeof(value_type), hipMemcpyDeviceToHost);
	string res;
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
			res += to_string(host_data[i * cols + j]) + " ";
		res += "\n";
	}
	if (IS_SAFE_DATA) memset(host_data, 0, static_cast<size_t>(rows) * cols * sizeof(value_type));
	delete[] host_data;
	return res;
}

template<typename Type>
void cumatrix<Type>::print() { print_matrix(); }

template<typename Type>
void cumatrix<Type>::resize(const uint32_t rows, const uint32_t cols) noexcept
{
	if (this->rows == rows && this->cols == cols)
		return;
	try
	{
		if (mat != nullptr)
			hipFree(mat);
		this->rows = rows;
		this->cols = cols;
		hipError_t err = hipMalloc((void**)&mat, static_cast<size_t>(rows) * cols * sizeof(value_type));
		err += hipMemset(mat, 0, static_cast<size_t>(rows) * cols * sizeof(value_type));
		if (err != hipSuccess)
			throw runtime_error("Failed to resize matrix.");
	}
	catch (const exception& err) {
		cerr << err.what() << endl;
	}
}

template<typename Type>
void cumatrix<Type>::resize(const uint32_t size) noexcept { resize(size, size); }

template<typename Type>
void cumatrix<Type>::update_dimensions(const uint32_t rows, const uint32_t cols)
{
	if (this->rows * this->cols != rows * cols)
		throw runtime_error("The number of elements in the matrix does not match the new shape.");
	this->rows = rows;
	this->cols = cols;
}

template<typename Type>
void cumatrix<Type>::update_dimensions(const uint32_t size) { update_dimensions(size, size); }

template<typename Type>
void cumatrix<Type>::reshape(const uint32_t rows, const uint32_t cols)
{
	if (this->rows == rows && this->cols == cols)
		return;
	pointer tmp = nullptr;
	hipMalloc((void**)&tmp, static_cast<size_t>(rows) * cols * sizeof(value_type));
	hipMemset(tmp, 0, static_cast<size_t>(rows) * cols * sizeof(value_type));
	dim3 blockSize = autoSetBlockSize2D(reshape_kernel<value_type>);
	dim3 gridSize = dim3((rows + blockSize.x - 1) / blockSize.x, (cols + blockSize.y - 1) / blockSize.y);
	reshape_kernel<value_type> << <gridSize, blockSize >> >
		(mat, tmp, rows, cols, row, col);
	hipDeviceSynchronize();
	hipFree(mat);
	mat = tmp;
	this->rows = row;
	this->cols = col;
	tmp = nullptr;
}

template<typename Type>
void cumatrix<Type>::reshape(const uint32_t size) { reshape(size, size); }

template<typename Type>
cumatrix<Type>::ElementProxy cumatrix<Type>::operator()(uint32_t row, uint32_t col)
{
	if (row >= rows || col >= cols)
		throw out_of_range("Index out of range.");
	return ElementProxy(*this, row, col);
}

template<typename Type>
void cumatrix<Type>::allocate_matrix(const uint32_t rows, const uint32_t cols)
{
	if (mat != nullptr)
		hipFree(mat);
	this->rows = rows;
	this->cols = cols;
	mat = allocator.allocate(static_cast<typename cuda_allocator<value_type>::size_type>(rows) * cols);
}

template<typename Type>
void cumatrix<Type>::deallocate_matrix() noexcept
{
	if (mat != nullptr)
		allocator.deallocate(mat, static_cast<typename cuda_allocator<value_type>::size_type>(rows) * cols);
	rows = 0;
	cols = 0;
	mat = nullptr;
}

template<typename Type>
uint32_t cumatrix<Type>::rowcount() const { return rows; }

template<typename Type>
uint32_t cumatrix<Type>::colcount() const { return cols; }

template<typename Type>
void cumatrix<Type>::get_data(pointer dst) const { hipMemcpy(dst, mat, _msize(dst) * sizeof(value_type), hipMemcpyDeviceToHost); }

template<typename Type>
void cumatrix<Type>::get_data(vector<value_type>& dst) const { hipMemcpy(dst.data(), mat, dst.size() * sizeof(value_type), hipMemcpyDeviceToHost); }

template<typename Type>
void cumatrix<Type>::get_data(vector<value_type>& dst, bool is_safesize) const
{
	if (is_safesize && dst.size() < static_cast<size_t>(rows) * cols)
		dst.resize(static_cast<size_t>(rows) * cols);
	hipMemcpy(dst.data(), mat, static_cast<size_t>(rows) * cols * sizeof(Type), hipMemcpyDeviceToHost);
}

template<typename Type>
void cumatrix<Type>::set_data(const pointer src) { hipMemcpy(mat, src, _msize(src) * sizeof(value_type), hipMemcpyHostToDevice); }

template<typename Type>
void cumatrix<Type>::set_data(const vector<value_type>& src) { hipMemcpy(mat, src.data(), src.size(), hipMemcpyHostToDevice); }

template<typename Type>
Type cumatrix<Type>::get(const uint32_t row, const uint32_t col) const
{
	if (row >= rows || col >= cols)
		throw out_of_range("Index out of range.");
	value_type res = 0;
	hipMemcpy(&res, mat + row * cols + col, sizeof(value_type), hipMemcpyDeviceToHost);
	return res;
}

template<typename Type>
cumatrix<Type>::ElementProxy::~ElementProxy()
{
	row = 0;
	col = 0;
}

template<typename Type>
cumatrix<Type>::ElementProxy::operator Type() { return mat.get(row, col); }

template<typename Type>
cumatrix<Type>::ElementProxy& cumatrix<Type>::ElementProxy::operator=(Type value)
{
	mat.set(row, col, value);
	return *this;
}
