#include "hip/hip_runtime.h"
#include "kernel_function.cuh"

template <typename Type>
__global__ static void identity_matrix_kernel(Type* data, const int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		data[idx * size + idx] = (Type)1;
}

template <typename Type>
__global__ static void ones_matrix_kernel(Type* data, const int total_elements)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < total_elements)
		data[idx] = (Type)1;
}

template <typename Type>
__global__ static void random_matrix_kernel
(Type* data, const int total_elements, hiprandState* states)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < total_elements)
		if constexpr (is_floating_point<Type>::value)
			data[idx] = hiprand_uniform(states + idx);
		else if constexpr (is_same<Type, double>::value)
			data[idx] = hiprand_uniform_double(states + idx);
		else
			data[idx] = (Type)hiprand(states + idx);
}

__global__ static void setup_random_kernel
(hiprandState* state, size_t seed, const int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		hiprand_init(seed, idx, 0, &state[idx]);
}

template <typename Type>
__global__ static void col_vec_broadcast_kernel
(const Type* src_vec, Type* res, const int size, const int cols)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		int row = idx / cols;
		res[idx] = src_vec[row];
	}
}

template <typename Type>
__global__ static void row_vec_broadcast_kernel
(const Type* src_vec, Type* res, const int size, const int cols)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		int col = idx % cols;
		res[idx] = src_vec[col];
	}
}

template <typename T1, typename T2, typename T3>
__global__ static void elementwise_add_kernel
(const T1* src1, const T2* src2, T3* res, const int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		res[idx] = (T3)(src1[idx] + src2[idx]);
}

template <typename T1, typename T2, typename T3>
__global__ static void elementwise_subtract_kernel
(const T1* src1, const T2* src2, T3* res, const int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		res[idx] = (T3)(src1[idx] - src2[idx]);
}

template <typename T1, typename T2, typename T3>
__global__ static void elementwise_multiply_kernel
(const T1* src1, const T2* src2, T3* res, const int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		res[idx] = (T3)(src1[idx] * src2[idx]);
}

template <typename T1, typename T2, typename T3>
__global__ static void elementwise_divide_kernel
(const T1* src1, const T2* src2, T3* res, const int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		res[idx] = (T3)(src1[idx] / src2[idx]);
}

template <typename T1, typename T2, typename T3>
__global__ static void matrix_scalar_add_kernel
(const T1* src, const T2 scalar, T3* res, const int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		res[idx] = (T3)(src[idx] + scalar);
}

template <typename T1, typename T2, typename T3>
__global__ static void matrix_scalar_subtract_kernel
(const T1* src, const T2 scalar, T3* res, const int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		res[idx] = (T3)(src[idx] - scalar);
}

template <typename T1, typename T2, typename T3>
__global__ static void matrix_scalar_multiply_kernel
(const T1* src, const T2 scalar, T3* res, const int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		res[idx] = (T3)(src[idx] * scalar);
}

template <typename T1, typename T2, typename T3>
__global__ static void matrix_scalar_divide_kernel
(const T1* src, const T2 scalar, T3* res, const int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		res[idx] = (T3)(src[idx] / scalar);
}

template <typename T1, typename T2, typename T3>
__global__ static void matrix_multiply_kernel
(const T1* src1, const T2* src2, T3* res, const int rows1, const int cols1, const int cols2)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = blockIdx.z * blockDim.z + threadIdx.z;

	if (row < rows1 && col < cols2)
	{
		T3 sum = 0;
		for (int i = 0; i < cols1; i++)
			sum += src1[row * cols1 + i] * src2[i * cols2 + col];
		res[row * cols2 + col] = sum;
	}
}