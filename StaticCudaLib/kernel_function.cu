#include "hip/hip_runtime.h"
﻿#pragma once
#include "kernel_function.cuh"

template <typename Type>
__global__ static void identity_matrix_kernel(Type* data, const uint32_t size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		data[idx * size + idx] = (Type)1;
}

template <typename Type>
__global__ static void ones_matrix_kernel(Type* data, const uint32_t total_elements)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < total_elements)
		data[idx] = (Type)1;
}

__global__ static void float_random_matrix_kernel
(float* data, const uint32_t total_elements, hiprandStatePhilox4_32_10_t* states)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < total_elements)
		data[idx] = hiprand_uniform(&states[idx]);
}

__global__ static void float_qrandom_matrix_kernel
(float* data, hiprandStateScrambledSobol32_t* states, const uint32_t n, const uint32_t dimensions)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n)
		for (int d = 0; d < dimensions; d++)
			data[idx * dimensions + d] = hiprand_uniform(&states[idx * dimensions + d]);
}

__global__ static void double_random_matrix_kernel
(double* data, const uint32_t total_elements, hiprandStatePhilox4_32_10_t* states)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < total_elements)
		data[idx] = hiprand_uniform_double(&states[idx]);
}

__global__ static void double_qrandom_matrix_kernel
(double* data, hiprandStateScrambledSobol64_t* states, const uint32_t n, const uint32_t dimensions)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n)
		for (int d = 0; d < dimensions; d++)
			data[idx * dimensions + d] = hiprand_uniform_double(&states[idx * dimensions + d]);
}

__global__ static void int_random_matrix_kernel
(int* data, const uint32_t total_elements, hiprandStatePhilox4_32_10_t* states)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < total_elements)
		data[idx] = hiprand(&states[idx]);
}

__global__ static void int_qrandom_matrix_kernel
(int* data, hiprandStateScrambledSobol32_t* states, const uint32_t n, const uint32_t dimensions)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n)
		for (int d = 0; d < dimensions; d++)
			data[idx * dimensions + d] = hiprand(&states[idx * dimensions + d]);
}

__global__ static void setup_random_kernel
(hiprandStatePhilox4_32_10_t* states, size_t seed, const uint32_t size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		hiprand_init(seed, idx, 0, states + idx);
}

__global__ static void setup_q32random_kernel
(hiprandStateScrambledSobol32_t* states, hiprandDirectionVectors32_t* dr_vec, const uint32_t n, const uint32_t dimensions)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n)
		for (int d = 0; d < dimensions; d++)
			hiprand_init(dr_vec[d], idx, 0, states + (idx * dimensions + d));
}

__global__ static void setup_q64random_kernel
(hiprandStateScrambledSobol64_t* states, hiprandDirectionVectors64_t* dr_vec, const uint32_t n, const uint32_t dimensions)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n)
		for (int d = 0; d < dimensions; d++)
			hiprand_init(dr_vec[d], idx, 0, states + (idx * dimensions + d));
}

template<typename Type> __global__ static void matrix_transpose_kernel
(const Type* src, Type* res, const uint32_t rows, const uint32_t cols)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx < rows && idy < cols)
		res[idy * rows + idx] = src[idx * cols + idy];
}

template <typename Type> __global__ void elementwise_equal_kernel
(const Type* src1, const Type* src2, bool* res, const uint32_t size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		res[idx] = (src1[idx] == src2[idx]);
}

template <typename Type> __global__ static void col_vec_broadcast_kernel
(const Type* src_vec, Type* res, const uint32_t size, const uint32_t cols)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		int row = idx / cols;
		res[idx] = src_vec[row];
	}
}

template <typename Type> __global__ static void row_vec_broadcast_kernel
(const Type* src_vec, Type* res, const uint32_t size, const uint32_t cols)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		int col = idx % cols;
		res[idx] = src_vec[col];
	}
}

template <typename T1, typename T2, typename T3>
__global__ static void elementwise_add_kernel
(const T1* src1, const T2* src2, T3* res, const uint32_t size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		res[idx] = (T3)(src1[idx] + src2[idx]);
}

template <typename T1, typename T2, typename T3>
__global__ static void elementwise_subtract_kernel
(const T1* src1, const T2* src2, T3* res, const uint32_t size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		res[idx] = (T3)(src1[idx] - src2[idx]);
}

template <typename T1, typename T2, typename T3>
__global__ static void elementwise_multiply_kernel
(const T1* src1, const T2* src2, T3* res, const uint32_t size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		res[idx] = (T3)(src1[idx] * src2[idx]);
}

template <typename T1, typename T2, typename T3>
__global__ static void elementwise_divide_kernel
(const T1* src1, const T2* src2, T3* res, const uint32_t size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		res[idx] = (T3)(src1[idx] / src2[idx]);
}

template <typename T1, typename T2, typename T3>
__global__ static void matrix_scalar_add_kernel
(const T1* src, const T2 scalar, T3* res, const uint32_t size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		res[idx] = (T3)(src[idx] + scalar);
}

template <typename T1, typename T2, typename T3>
__global__ static void matrix_scalar_subtract_kernel
(const T1* src, const T2 scalar, T3* res, const uint32_t size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		res[idx] = (T3)(src[idx] - scalar);
}

template <typename T1, typename T2, typename T3>
__global__ static void matrix_scalar_multiply_kernel
(const T1* src, const T2 scalar, T3* res, const uint32_t size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		res[idx] = (T3)(src[idx] * scalar);
}

template <typename T1, typename T2, typename T3>
__global__ static void matrix_scalar_divide_kernel
(const T1* src, const T2 scalar, T3* res, const uint32_t size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		res[idx] = (T3)(src[idx] / scalar);
}

template <typename T1, typename T2, typename T3>
__global__ static void matrix_multiply_kernel
(const T1* src1, const T2* src2, T3* res, const uint32_t rows1, const uint32_t cols1, const uint32_t cols2)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = blockIdx.z * blockDim.z + threadIdx.z;

	if (row < rows1 && col < cols2)
	{
		T3 sum = 0;
		for (int i = 0; i < cols1; i++)
			sum += src1[row * cols1 + i] * src2[i * cols2 + col];
		res[row * cols2 + col] = sum;
	}
}

template <typename T> __global__ static void reshape_kernel
(const T* src, T* res, const uint32_t rows_old, const uint32_t cols_old,const uint32_t rows_new, const uint32_t cols_new)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	if (idx < rows_new && idy < cols_new && idx < rows_old && idy < cols_old)
	{
		res[idx * cols_new + idy] = src[idx * cols_old + idy];
	}
}