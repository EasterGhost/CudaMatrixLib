#include "hip/hip_runtime.h"
﻿#pragma once

#include "kernel_function.cu"
#include "kernel_function.cuh"
#include "TemplateCudaMatrix.cuh"

extern clock_t time_used_init = 0;
extern clock_t time_used_gen_init = 0;
extern clock_t time_used_gen = 0;
extern clock_t time_used_switch_type = 0;
extern clock_t time_used_setblock = 0;
extern clock_t time_used_end = 0;

template <class T>
static int autoSetBlockSize(T func)
{
	int blockSize = 0;
	int gridSize = 0;
	hipOccupancyMaxPotentialBlockSize(&gridSize, &blockSize, func, 0, 0);
	if (blockSize < 32)
		blockSize = 32;
	if (blockSize == 0)
		throw runtime_error("Failed to set block size.");
	return blockSize;
}

template <class T>
static dim3 autoSetBlockSize2D(T func, const int rows, const int cols)
{
	int blockSize = 0;
	int gridSize = 0;
	hipOccupancyMaxPotentialBlockSize(&gridSize, &blockSize, func, 0, 0);
	if (blockSize == 0)
		throw runtime_error("Failed to set block size.");
	return dim3(blockSize, 1);
}

template <typename Type>
CudaMatrix<Type>::CudaMatrix()
{
	rows = 0;
	cols = 0;
	mat = nullptr;
}

template <typename Type>
CudaMatrix<Type>::CudaMatrix(const unsigned int rows, const unsigned int cols) : rows(rows), cols(cols)
{
	int total_elements = rows * cols;
	hipMalloc((void**)&mat, total_elements * sizeof(Type));
	hipMemset(mat, 0, total_elements * sizeof(Type));
	hipblasCreate(&handle);
	hipsolverDnCreate(&solver_handle);
}

template <typename Type>
CudaMatrix<Type>::CudaMatrix(const unsigned int rows, const unsigned int cols, const MatrixType type) : rows(rows), cols(cols)
{
	clock_t start = clock();
	size_t total_elements = static_cast<size_t>(rows) * cols;
	hipMalloc((void**)&mat, total_elements * sizeof(Type));
	int blockSize = 0;
	int gridSize = 0;
	hiprandStatePhilox4_32_10_t* states = nullptr;
	hiprandStateScrambledSobol32_t* qstates32 = nullptr;
	hiprandStateScrambledSobol64_t* qstates64 = nullptr;
	hiprandDirectionVectors32_t* dr_vec32 = nullptr;
	hiprandDirectionVectors64_t* dr_vec64 = nullptr;
	hipDeviceSynchronize();
	time_used_init += clock() - start;
	switch (type)
	{
	case Zero:
		hipMemset(mat, 0, total_elements * sizeof(Type));
		break;
	case Ones:
		blockSize = autoSetBlockSize(ones_matrix_kernel<Type>);
		gridSize = (total_elements + blockSize - 1) / blockSize;
		ones_matrix_kernel<Type> << <gridSize, blockSize >> > (mat, total_elements);
		hipDeviceSynchronize();
		break;
	case Identity:
		if (rows != cols)
			throw runtime_error("Identity matrix must be square matrix.");
		hipMemset(mat, 0, static_cast<size_t>(rows) * cols * sizeof(Type));
		blockSize = autoSetBlockSize(identity_matrix_kernel<Type>);
		gridSize = (rows + blockSize - 1) / blockSize;
		identity_matrix_kernel<Type> << <gridSize, blockSize >> > (mat, rows);
		hipDeviceSynchronize();
		break;
	case Random:
		start = clock();
		blockSize = autoSetBlockSize(setup_random_kernel);
		//cout << "Block size of setup random kernel: " << blockSize << endl;
		gridSize = (total_elements + blockSize - 1) / blockSize;
		hipMalloc((void**)&states, total_elements * sizeof(hiprandStatePhilox4_32_10_t));
		setup_random_kernel << <gridSize, blockSize >> > (states, time(0), total_elements);
		hipDeviceSynchronize();
		time_used_gen_init += clock() - start;
		start = clock();
		if constexpr (is_same<Type, float>::value)
		{
			time_used_switch_type += clock() - start;
			start = clock();
			blockSize = autoSetBlockSize(float_random_matrix_kernel);
			//cout << "Block size of float random kernel: " << blockSize << endl;
			gridSize = (total_elements + blockSize - 1) / blockSize;
			time_used_setblock += clock() - start;
			start = clock();
			float_random_matrix_kernel << <gridSize, blockSize >> >
				((float*)mat, total_elements, states);
			hipDeviceSynchronize();
			time_used_gen += clock() - start;
			start = clock();
		}
		else if constexpr (is_same<Type, double>::value)
		{
			time_used_switch_type += clock() - start;
			start = clock();
			blockSize = autoSetBlockSize(double_random_matrix_kernel);
			//cout << "Block size of double random kernel: " << blockSize << endl;
			gridSize = (total_elements + blockSize - 1) / blockSize;
			time_used_setblock += clock() - start;
			start = clock();
			double_random_matrix_kernel << <gridSize, blockSize >> >
				((double*)mat, total_elements, states);
			hipDeviceSynchronize();
			time_used_gen += clock() - start;
			start = clock();
		}
		else if constexpr (is_same<Type, int>::value)
		{
			time_used_switch_type += clock() - start;
			start = clock();
			blockSize = autoSetBlockSize(int_random_matrix_kernel);
			//cout << "Block size of int random kernel: " << blockSize << endl;
			gridSize = (total_elements + blockSize - 1) / blockSize;
			time_used_setblock += clock() - start;
			start = clock();
			int_random_matrix_kernel << <gridSize, blockSize >> >
				((int*)mat, total_elements, states);
			hipDeviceSynchronize();
			time_used_gen += clock() - start;
			start = clock();
		}
		else
		{
			total_elements = static_cast<size_t>(rows) * cols * sizeof(Type) / sizeof(int);
			//hipFree(states);
			hipMalloc((void**)&states, total_elements * sizeof(hiprandStatePhilox4_32_10_t));
			setup_random_kernel << <gridSize, blockSize >> > (states, time(0), total_elements);
			hipDeviceSynchronize();
			time_used_gen_init += clock() - start;
			start = clock();
			blockSize = autoSetBlockSize(int_random_matrix_kernel);
			//cout << "Block size of int random kernel: " << blockSize << endl;
			gridSize = (total_elements + blockSize - 1) / blockSize;
			time_used_setblock += clock() - start;
			start = clock();
			int_random_matrix_kernel << <gridSize, blockSize >> >
				((int*)mat, total_elements, states);
			hipDeviceSynchronize();
			time_used_gen += clock() - start;
			start = clock();
		}
		hipFree(states);
		break;
	case QuasiRandom:
		start = clock();
		if constexpr (is_same<Type, float>::value)
		{
			blockSize = autoSetBlockSize(setup_q32random_kernel);
			//cout << "Block size of setup random kernel: " << blockSize << endl;
			gridSize = (total_elements + blockSize - 1) / blockSize;
			hipMalloc((void**)&qstates32, total_elements * sizeof(hiprandStateScrambledSobol32_t));
			hipMalloc((void**)&dr_vec32, cols * sizeof(hiprandDirectionVectors32_t));
			hiprandGetDirectionVectors32(&dr_vec32, HIPRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6);
			setup_q32random_kernel << <gridSize, blockSize >> > (qstates32, dr_vec32, rows, cols);
			hipDeviceSynchronize();
			time_used_gen_init += clock() - start;
			start = clock();
			blockSize = autoSetBlockSize(float_qrandom_matrix_kernel);
			//cout << "Block size of float random kernel: " << blockSize << endl;
			gridSize = (total_elements + blockSize - 1) / blockSize;
			time_used_setblock += clock() - start;
			start = clock();
			float_qrandom_matrix_kernel << <gridSize, blockSize >> >((float*)mat, qstates32, rows, cols);
			hipDeviceSynchronize();
			time_used_gen += clock() - start;
			start = clock();
			hipFree(qstates32);
			hipFree(dr_vec32);
		}
		else if constexpr (is_same<Type, double>::value)
		{
			blockSize = autoSetBlockSize(setup_q64random_kernel);
			//cout << "Block size of setup random kernel: " << blockSize << endl;
			gridSize = (total_elements + blockSize - 1) / blockSize;
			hipMalloc((void**)&qstates64, total_elements * sizeof(hiprandStateScrambledSobol64_t));
			hipMalloc((void**)&dr_vec64, cols * sizeof(hiprandDirectionVectors64_t));
			hiprandGetDirectionVectors64(&dr_vec64, HIPRAND_SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6);
			setup_q64random_kernel << <gridSize, blockSize >> > (qstates64, dr_vec64, rows, cols);
			hipDeviceSynchronize();
			time_used_gen_init += clock() - start;
			start = clock();
			blockSize = autoSetBlockSize(double_qrandom_matrix_kernel);
			//cout << "Block size of double random kernel: " << blockSize << endl;
			gridSize = (total_elements + blockSize - 1) / blockSize;
			time_used_setblock += clock() - start;
			start = clock();
			double_qrandom_matrix_kernel << <gridSize, blockSize >> >
				((double*)mat, qstates64, rows, cols);
			hipDeviceSynchronize();
			time_used_gen += clock() - start;
			start = clock();
			hipFree(qstates64);
			hipFree(dr_vec64);
		}
		else if constexpr (is_same<Type, int>::value)
		{
			blockSize = autoSetBlockSize(setup_q32random_kernel);
			//cout << "Block size of setup random kernel: " << blockSize << endl;
			//system("pause");
			gridSize = (total_elements + blockSize - 1) / blockSize;
			hipMalloc((void**)&qstates32, total_elements * sizeof(hiprandStateScrambledSobol32_t));
			hipMalloc((void**)&dr_vec32, cols * sizeof(hiprandDirectionVectors32_t));
			hiprandGetDirectionVectors32(&dr_vec32, HIPRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6);
			setup_q32random_kernel << <gridSize, blockSize >> > (qstates32, dr_vec32, rows, cols);
			hipDeviceSynchronize();
			time_used_gen_init += clock() - start;
			start = clock();
			blockSize = autoSetBlockSize(int_qrandom_matrix_kernel);
			cout << "Block size of int random kernel: " << blockSize << endl;
			gridSize = (total_elements + blockSize - 1) / blockSize;
			time_used_setblock += clock() - start;
			start = clock();
			int_qrandom_matrix_kernel << <gridSize, blockSize >> >((int*)mat, qstates32, rows, cols);
			hipDeviceSynchronize();
			time_used_gen += clock() - start;
			start = clock();
			hipFree(qstates32);
			hipFree(dr_vec32);
		}
		else
		{
			total_elements = static_cast<size_t>(rows) * cols * sizeof(Type) / sizeof(int);
			hipMalloc((void**)&qstates32, total_elements * sizeof(hiprandStateScrambledSobol32_t));
			hipMalloc((void**)&dr_vec32, cols * sizeof(hiprandDirectionVectors32_t));
			hiprandGetDirectionVectors32(&dr_vec32, HIPRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6);
			setup_q32random_kernel << <gridSize, blockSize >> > (qstates32, dr_vec32, rows, cols);
			hipDeviceSynchronize();
			time_used_gen_init += clock() - start;
			start = clock();
			blockSize = autoSetBlockSize(int_qrandom_matrix_kernel);
			//cout << "Block size of int random kernel: " << blockSize << endl;
			gridSize = (total_elements + blockSize - 1) / blockSize;
			time_used_setblock += clock() - start;
			start = clock();
			int_qrandom_matrix_kernel << <gridSize, blockSize >> >
				((int*)mat, qstates32, rows, cols);
			hipDeviceSynchronize();
			time_used_gen += clock() - start;
			start = clock();
			hipFree(qstates32);
			hipFree(dr_vec32);
		}
		break;
	default:
		throw runtime_error("Unknown matrix type.");
	}
	hipDeviceSynchronize();
	hipblasCreate(&handle);
	hipsolverDnCreate(&solver_handle);
	time_used_end += clock() - start;
}

template <typename Type>
CudaMatrix<Type>::CudaMatrix(const unsigned int size) : CudaMatrix(size, size) {}

template <typename Type>
CudaMatrix<Type>::CudaMatrix(const unsigned int size, const MatrixType type) : CudaMatrix(size, size, type) {}

template <typename Type>
CudaMatrix<Type>::CudaMatrix(const unsigned int rows, const unsigned int cols, const Type* src) : CudaMatrix(rows, cols) { hipMemcpy(mat, src, static_cast<size_t>(rows) * cols * sizeof(Type), hipMemcpyHostToDevice); }

template <typename Type>
CudaMatrix<Type>::CudaMatrix(const unsigned int size, const Type* src) : CudaMatrix(size, size, src) {}

template<typename Type>
CudaMatrix<Type>::CudaMatrix(const unsigned int size, const vector<Type>& src) : CudaMatrix(size, size, src.data()) {}

template <typename Type>
CudaMatrix<Type>::CudaMatrix(const unsigned int rows, unsigned int cols, const vector<Type>& src) : CudaMatrix(rows, cols, src.data()) {}

template <typename Type>
CudaMatrix<Type>::CudaMatrix(const CudaMatrix<Type>& other) : rows(other.rows), cols(other.cols)
{
	int total_elements = rows * cols;
	hipMalloc((void**)&mat, total_elements * sizeof(Type));
	hipMemcpy(mat, other.mat, total_elements * sizeof(Type), hipMemcpyDeviceToDevice);
	hipblasCreate(&handle);
	hipsolverDnCreate(&solver_handle);
}

template <typename Type>
CudaMatrix<Type>::~CudaMatrix()
{
	hipMemset(mat, 0, static_cast<size_t>(rows) * cols * sizeof(Type));
	hipFree(mat);
	hipblasDestroy(handle);
	hipsolverDnDestroy(solver_handle);
	rows = 0;
	cols = 0;
	mat = nullptr;
}

template<typename Type>
void CudaMatrix<Type>::set(const unsigned int row, const unsigned int col, const Type value)
{
	if (row >= rows || col >= cols)
		throw out_of_range("Index out of range.");
	CUDA_CHECK(hipMemcpy(mat + row * cols + col, &value, sizeof(Type), hipMemcpyHostToDevice));
}

template<typename Type>
Type* CudaMatrix<Type>::data() const { return this->mat; }

template<typename Type>
void CudaMatrix<Type>::print()
{
	Type* host_data = new Type[rows * cols];
	hipMemcpy(host_data, mat, static_cast<size_t>(rows) * cols * sizeof(Type), hipMemcpyDeviceToHost);
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
			cout << host_data[i * cols + j] << " ";
		cout << endl;
	}
	delete[] host_data;
}

template<typename Type>
unsigned int CudaMatrix<Type>::getRows() const { return rows; }

template<typename Type>
unsigned int CudaMatrix<Type>::getCols() const { return cols; }

template<typename Type>
void CudaMatrix<Type>::getData(Type* dst) const { hipMemcpy(dst, mat, static_cast<size_t>(rows) * cols * sizeof(Type), hipMemcpyDeviceToHost); }

template<typename Type>
void CudaMatrix<Type>::setData(const vector<Type>& src) { hipMemcpy(mat, src.data(), src.size(), hipMemcpyHostToDevice); }

template<typename Type>
Type CudaMatrix<Type>::get(const unsigned int row, const unsigned int col) const
{
	if (row >= rows || col >= cols)
		throw out_of_range("Index out of range.");
	Type res = 0;
	hipMemcpy(&res, mat + row * cols + col, sizeof(Type), hipMemcpyDeviceToHost);
	return res;
}

template<typename Type>
template<typename T>
void CudaMatrix<Type>::add(const CudaMatrix<T>& other)
{
	if (rows != other.rows || cols != other.cols)
		throw runtime_error("Matrix size does not match.");
	int total_elements = rows * cols;
	int blockSize = autoSetBlockSize(elementwise_add_kernel<Type, T, Type>);
	int gridSize = (total_elements + blockSize - 1) / blockSize;
	elementwise_add_kernel<Type, T, Type> << <gridSize, blockSize >> >
		(mat, other.mat, mat, total_elements);
}