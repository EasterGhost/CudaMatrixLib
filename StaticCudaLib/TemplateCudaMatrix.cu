#include "TemplateCudaMatrix.cuh"
#include "kernel_function.cuh"
#include "kernel_function.cu"

template <class T>
static int autoSetBlockSize(T func)
{
	int blockSize = 0;
	int gridSize = 0;
	hipOccupancyMaxPotentialBlockSize(&gridSize, &blockSize, func, 0, 0);
	if (blockSize == 0)
		throw runtime_error("Failed to set block size.");
	return blockSize;
}

template <class T>
static dim3 autoSetBlockSize2D(T func, const int rows, const int cols)
{
	int blockSize = 0;
	int gridSize = 0;
	hipOccupancyMaxPotentialBlockSize(&gridSize, &blockSize, func, 0, 0);
	if (blockSize == 0)
		throw runtime_error("Failed to set block size.");
	return dim3(blockSize, 1);
}

template <typename Type>
CudaMatrix<Type>::CudaMatrix()
{
	rows = 0;
	cols = 0;
	mat = nullptr;
}

template <typename Type>
CudaMatrix<Type>::CudaMatrix(int rows, int cols) : rows(rows), cols(cols)
{
	int total_elements = rows * cols;
	hipMalloc((void**)&mat, total_elements * sizeof(Type));
	hipMemset(mat, 0, total_elements * sizeof(Type));
	hipblasCreate(&handle);
	hipsolverDnCreate(&solver_handle);
}

template <typename Type>
CudaMatrix<Type>::CudaMatrix(int rows, int cols, MatrixType type) : rows(rows), cols(cols)
{
	int total_elements = rows * cols;
	hipMalloc((void**)&mat, total_elements * sizeof(Type));
	int blockSize = 0;
	int gridSize = 0;
	hiprandState* states = nullptr;

	switch (type)
	{
	case Zero:
		hipMemset(mat, 0, total_elements * sizeof(Type));
		break;
	case Ones:
		blockSize = autoSetBlockSize(ones_matrix_kernel<Type>);
		gridSize = (total_elements + blockSize - 1) / blockSize;
		ones_matrix_kernel<Type> << <gridSize, blockSize >> > (mat, total_elements);
		break;
	case Identity:
		if (rows != cols)
		{
			throw runtime_error("Identity matrix must be square matrix.");
		}
		hipMemset(mat, 0, static_cast<size_t>(rows) * cols * sizeof(Type));
		blockSize = autoSetBlockSize(identity_matrix_kernel<Type>);
		gridSize = (rows + blockSize - 1) / blockSize;
		identity_matrix_kernel<Type> << <gridSize, blockSize >> > (mat, rows);
		break;
	case Random:
		blockSize = autoSetBlockSize(random_matrix_kernel<Type>);
		gridSize = (total_elements + blockSize - 1) / blockSize;
		hipMalloc((void**)&states, total_elements * sizeof(hiprandState));
		setup_random_kernel << <gridSize, blockSize >> > (states, time(0), total_elements);
		random_matrix_kernel<Type> << <gridSize, blockSize >> > (mat, total_elements, states);

		break;
	default:
		throw runtime_error("Unknown matrix type.");
	}
	hipblasCreate(&handle);
	hipsolverDnCreate(&solver_handle);
	hipFree(states);
}

template <typename Type>
CudaMatrix<Type>::CudaMatrix(int size) : CudaMatrix(size, size) {}

template <typename Type>
CudaMatrix<Type>::CudaMatrix(int size, MatrixType type) : CudaMatrix(size, size, type) {}

template <typename Type>
CudaMatrix<Type>::CudaMatrix(int rows, int cols, Type* src) : CudaMatrix(rows, cols) { hipMemcpy(mat, src, static_cast<size_t>(rows) * cols * sizeof(Type), hipMemcpyHostToDevice); }

template <typename Type>
CudaMatrix<Type>::CudaMatrix(int size, Type* src) : CudaMatrix(size, size, src) {}

template<typename Type>
CudaMatrix<Type>::CudaMatrix(int size, vector<Type> src) : CudaMatrix(size, size, src.data()) {}

template <typename Type>
CudaMatrix<Type>::CudaMatrix(int rows, int cols, vector<Type> src) : CudaMatrix(rows, cols, src.data()) {}

template <typename Type>
CudaMatrix<Type>::CudaMatrix(const CudaMatrix<Type>& other) : rows(other.rows), cols(other.cols)
{
	int total_elements = rows * cols;
	hipMalloc((void**)&mat, total_elements * sizeof(Type));
	hipMemcpy(mat, other.mat, total_elements * sizeof(Type), hipMemcpyDeviceToDevice);
	hipblasCreate(&handle);
	hipsolverDnCreate(&solver_handle);
}

template <typename Type>
CudaMatrix<Type>::~CudaMatrix()
{
	hipMemset(mat, 0, static_cast<size_t>(rows) * cols * sizeof(Type));
	rows = 0;
	cols = 0;
	hipFree(mat);
	mat = nullptr;
	hipblasDestroy(handle);
	hipsolverDnDestroy(solver_handle);
}

template<typename Type>
void CudaMatrix<Type>::set(const int row, const int col, const Type value)
{
	if (row < 0 || row >= rows || col < 0 || col >= cols)
		throw out_of_range("Index out of range.");
	CUDA_CHECK(hipMemcpy(mat + row * cols + col, &value, sizeof(Type), hipMemcpyHostToDevice));
}

template<typename Type>
Type* CudaMatrix<Type>::data() const { return this->mat; }

template<typename Type>
void CudaMatrix<Type>::print()
{
	Type* host_data = new Type[rows * cols];
	CUDA_CHECK(hipMemcpy(host_data, mat, rows * cols * sizeof(Type), hipMemcpyDeviceToHost));
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
			cout << host_data[i * cols + j] << " ";
		cout << endl;
	}
	delete[] host_data;
}

template<typename Type>
int CudaMatrix<Type>::getRows() const { return rows; }

template<typename Type>
int CudaMatrix<Type>::getCols() const { return cols; }

template<typename Type>
void CudaMatrix<Type>::getData(Type* dst) const { hipMemcpy(dst, mat, static_cast<size_t>(rows) * cols * sizeof(Type), hipMemcpyDeviceToHost); }

template<typename Type>
void CudaMatrix<Type>::setData(const vector<Type>& src) { hipMemcpy(mat, src.data(), src.size(), hipMemcpyHostToDevice); }

template<typename Type>
Type CudaMatrix<Type>::get(const int row, const int col) const
{
	if (row < 0 || row >= rows || col < 0 || col >= cols)
		throw out_of_range("Index out of range.");
	Type res = 0;
	hipMemcpy(&res, mat + row * cols + col, sizeof(Type), hipMemcpyDeviceToHost);
	return res;
}

template<typename Type>
template<typename T>
void CudaMatrix<Type>::add(const CudaMatrix<T>& other)
{
	if (rows != other.rows || cols != other.cols)
		throw runtime_error("Matrix size does not match.");
	int total_elements = rows * cols;
	int blockSize = autoSetBlockSize(elementwise_add_kernel<Type, T, Type>);
	int gridSize = (total_elements + blockSize - 1) / blockSize;
	elementwise_add_kernel<Type, T, Type> << <gridSize, blockSize >> > (mat, other.mat, mat, total_elements);
}