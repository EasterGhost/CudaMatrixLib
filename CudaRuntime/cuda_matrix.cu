#include "hip/hip_runtime.h"
﻿/**
* @file cuda_matrix.cu
* @brief CUDA 矩阵类实现文件
* @note 仅支持 float 类型
* @note 使用 cuBLAS 实现矩阵运算
* @date 2024-12-16
* @version 1.0
* @author LiMuchen
* @license MIT
*/
#include "cuda_matrix.h"

extern "C" __global__ static void extractSubMatrixKernel(
	const float* __restrict__ d_data, // 源矩阵数据指针
	float* d_sub_data,                // 子矩阵数据指针
	//int src_rows,                   // 源矩阵行数
	int src_cols,                     // 源矩阵列数
	int start_row,                    // 子矩阵起始行索引
	int start_col,                    // 子矩阵起始列索引
	int sub_rows,                     // 子矩阵行数
	int sub_cols                      // 子矩阵列数
) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int total_elements = sub_rows * sub_cols;

	if (tid < total_elements) {
		// 计算子矩阵中的行列索引
		int sub_row = tid / sub_cols;
		int sub_col = tid % sub_cols;

		// 对应到源矩阵中的行列索引
		int src_row = start_row + sub_row;
		int src_col = start_col + sub_col;

		// 计算线性索引
		int src_idx = src_row * src_cols + src_col;
		int dst_idx = sub_row * sub_cols + sub_col;

		// 复制元素
		d_sub_data[dst_idx] = d_data[src_idx];
	}
}

extern "C" __global__ static void extractSubMatrixIndexedKernel(
	const float* __restrict d_data,			// 源矩阵数据指针
	float* d_sub_data,						// 子矩阵数据指针
	const int* __restrict d_row_indices,	// 子矩阵行索引
	const int* __restrict d_col_indices,	// 子矩阵列索引
	int src_cols,							// 源矩阵列数
	int sub_rows,							// 子矩阵行数
	int sub_cols							// 子矩阵列数
) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < sub_rows * sub_cols) {
		int sub_row = tid / sub_cols;
		int sub_col = tid % sub_cols;

		int src_row = d_row_indices[sub_row];
		int src_col = d_col_indices[sub_col];

		int src_idx = src_row * src_cols + src_col;
		int dst_idx = sub_row * sub_cols + sub_col;

		d_sub_data[dst_idx] = d_data[src_idx];
	}
}


extern "C" __global__ static void reshape_kernel(const float* data, float* result, int rows_old, int cols_old, int rows_new, int cols_new) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	if (idx < rows_new && idy < cols_new && idx < rows_old && idy < cols_old) {
		result[idx * cols_new + idy] = data[idx * cols_old + idy];
	}
}

extern "C" __global__ static void fill_diag_kernel(float* matrix, float* diag, int offset, int size) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size) {
		int col = idx + offset;
		int row = idx;
		if (col < size && col >= 0)
			matrix[row * size + col] = diag[idx];
	}
}

extern "C" __device__ static double atomicAdd_double(double* address, double val)
{
	unsigned long long int* address_as_ull =
		(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;

	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
			__double_as_longlong(val +
				__longlong_as_double(assumed)));

		// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
	} while (assumed != old);

	return __longlong_as_double(old);
}

extern "C" __global__ static void get_diag(const float* matrix, float* result, int size) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size) {
		result[idx] = (float)matrix[idx * size + idx];
	}
}

extern "C" __global__ static void reduce_sum(float* d_input, float* d_output, int n) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	float temp = 0.0;
	// 使用归约方法进行求和
	while (tid < n) {
		temp += d_input[tid];
		tid += stride;
	}

	// 将每个线程的结果存到共享内存
	extern __shared__ float shared_sum[512];
	int lane = threadIdx.x;

	shared_sum[lane] = temp;
	__syncthreads();

	// 归约：每个block内部合并
	for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
		if (lane < offset) {
			shared_sum[lane] += shared_sum[lane + offset];
		}
		__syncthreads();
	}

	// 最后一个线程将结果写到全局内存
	if (lane == 0) {
		atomicAdd(d_output, shared_sum[0]);
	}
}

extern "C" __global__ static void reduce_multi(float* d_input, float* d_output, int n) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	float temp = 1.0f;
	// 使用归约方法进行求和
	while (tid < n) {
		temp *= d_input[tid];
		tid += stride;
	}

	// 将每个线程的结果存到共享内存
	extern __shared__ float shared_sum[512];
	int lane = threadIdx.x;

	shared_sum[lane] = temp;
	__syncthreads();

	// 归约：每个block内部合并
	for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
		if (lane < offset) {
			shared_sum[lane] *= shared_sum[lane + offset];
		}
		__syncthreads();
	}

	// 最后一个线程将结果写到全局内存
	if (lane == 0) {
		atomicAdd(d_output, shared_sum[0]);
	}
}

extern "C" __global__ static void norm_kernel(const float* matrix, float* result, int size, int L) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) {
		result[idx] = (float)pow(fabsf(matrix[idx]), L);
	}
}

extern "C" __global__ static void divide_kernel(float* A, float* B, float* C, int size) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size) {
		C[idx] = A[idx] / B[idx];
	}
}

extern "C" __global__ static void random_kernel(hiprandState* state, unsigned long seed) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	hiprand_init(seed, idx, 0, &state[idx]);
}

extern "C" __global__ static void generate_random_numbers(hiprandState* globalState, float* data, int size) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size) {
		hiprandState localState = globalState[idx];
		data[idx] = hiprand_uniform(&localState);
		globalState[idx] = localState;
	}
}

extern "C" __global__ static void identity_matrix_kernel(float* data, int rows, int cols) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int total_elements = rows * cols;
	if (idx < total_elements) {
		int row = idx / cols;
		int col = idx % cols;
		if (row == col) { data[idx] = 1.0f; }
	}
}

extern "C" __global__ static void ones_matrix_kernel(float* data, int total_elements) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < total_elements) {
		data[idx] = 1.0f;
	}
}

extern "C" __global__ static void elementwise_multiply_kernel(const float* A, const float* B, float* C, int size) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size) {
		C[idx] = A[idx] * B[idx];
	}
}

extern "C" __global__ static void col_vec_broadcast2matrix_kernel(const float* sourceVector, float* resultMatrix, int cols, int size) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size) {
		int row = idx / cols;
		resultMatrix[idx] = sourceVector[row];
	}
}

extern "C" __global__ static void row_vec_broadcast2matrix_kernel(const float* sourceVector, float* resultMatrix, int cols, int size) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size) {
		int col = idx % cols;
		resultMatrix[idx] = sourceVector[col];
	}
}

extern "C" __global__ static void setdiff_kernel(const float* A, const float* B, float* result, int sizeA, int sizeB) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < sizeA) {
		bool found = false;
		for (int j = 0; j < sizeB; ++j) {
			if (A[idx] == B[j]) {
				found = true;
				break;
			}
		}
		if (!found) {
			result[idx] = A[idx];
		}
		else {
			result[idx] = NAN; // Use NaN to indicate that the element is not in the result
		}
	}
}

static void checkCudaError(hipError_t err, const char* msg) {
	if (err != hipSuccess) {
		std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
		system("pause");
		exit(EXIT_FAILURE);
	}
}

template<class T>
static int autoSetBlockSize(T func) {
	int blockSize = 0;
	int minGridSize = 0;
	hipError_t status = hipOccupancyMaxPotentialBlockSize(
		&minGridSize,	// 返回的最小网格大小
		&blockSize,		// 返回的最佳线程块大小
		func,			// 内核函数
		0,				// 动态共享内存大小
		0);				// 块大小限制
	if (status != hipSuccess) {
		throw runtime_error("hipOccupancyMaxPotentialBlockSize failed!" + string(hipGetErrorString(status)));
	}
	return blockSize;
}

template<class T>
static dim3 autoSetBlockSize2D(T func, int rows, int cols) {
	int blockSize = 0;
	int minGridSize = 0;
	hipError_t status = hipOccupancyMaxPotentialBlockSize(
		&minGridSize,  // 返回的最小网格大小
		&blockSize,    // 返回的最佳线程块大小
		func,          // 内核函数
		0,             // 动态共享内存大小
		0);            // 块大小限制
	if (status != hipSuccess) {
		throw runtime_error("hipOccupancyMaxPotentialBlockSize failed!" + string(hipGetErrorString(status)));
	}

	// 计算二维线程块的大小
	int blockDimX = sqrt(blockSize);
	int blockDimY = blockSize / blockDimX;

	// 确保线程块大小不超过矩阵的维度
	blockDimX = min(blockDimX, rows);
	blockDimY = min(blockDimY, cols);

	return dim3(blockDimX, blockDimY);
}

static void findMax(const vector<float>& data, int start, int end, float& max_value, mutex& mtx) {
	float local_max = -numeric_limits<float>::infinity();
	for (int i = start; i < end; ++i) {
		if (data[i] > local_max) {
			local_max = data[i];
		}
	}
	// 使用互斥锁保护对全局最大值的更新
	std::lock_guard<mutex> lock(mtx);
	if (local_max > max_value) {
		max_value = local_max;
	}
}

cudaMatrix::cudaMatrix() : rows(0), cols(0), data(nullptr) {}

cudaMatrix::cudaMatrix(int rows, int cols) : rows(rows), cols(cols) {
	hipError_t err = hipMalloc((void**)&data, static_cast<size_t>(rows) * cols * sizeof(float));
	if (err != hipSuccess) {
		throw runtime_error("hipMalloc failed! (code M0001)" + string(hipGetErrorString(err)));
	}
	hipMemset(&data, 0, static_cast<size_t>(rows) * cols * sizeof(float));
}

cudaMatrix::cudaMatrix(int size) : rows(size), cols(size) {
	hipError_t err = hipMalloc((void**)&data, static_cast<size_t>(rows) * cols * sizeof(float));
	if (err != hipSuccess) {
		throw runtime_error("hipMalloc failed! (code M0002)");
	}
	hipMemset(data, 0, static_cast<size_t>(rows) * cols * sizeof(float));
}

cudaMatrix::cudaMatrix(int rows, int cols, MatrixType type) : rows(rows), cols(cols) {
	int size = rows * cols;
	hipError_t err = hipMalloc((void**)&data, size * sizeof(float));
	if (err != hipSuccess) {
		throw runtime_error("hipMalloc failed! (code M0003)");
	}
	int threadsPerBlock = 0;
	int blocksPerGrid = 0;
	switch (type)
	{
	case Zero:
		hipMemset(data, 0, size * sizeof(float));
		break;
	case Ones:
		threadsPerBlock = autoSetBlockSize(ones_matrix_kernel);
		blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
		ones_matrix_kernel << <blocksPerGrid, threadsPerBlock >> > (data, size);
		break;
	case Identity:
		hipMemset(data, 0, size * sizeof(float));
		threadsPerBlock = autoSetBlockSize(identity_matrix_kernel);
		blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
		identity_matrix_kernel << <blocksPerGrid, threadsPerBlock >> > (data, rows, cols);
		break;
	case Random:
		hiprandState* state = nullptr;
		hipError_t err = hipMalloc((void**)&state, size * sizeof(hiprandState));
		if (err != hipSuccess) {
			throw runtime_error("hipMalloc failed for hiprandState! (code M0005)");
		}
		threadsPerBlock = autoSetBlockSize(random_kernel);
		blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
		random_kernel << <blocksPerGrid, threadsPerBlock >> > (state, (unsigned long)time(0));
		generate_random_numbers << <blocksPerGrid, threadsPerBlock >> > (state, data, size);
		hipFree(state);
		break;
	}
}

cudaMatrix::cudaMatrix(int size, MatrixType type) : rows(size), cols(size) {
	int total_elements = rows * cols;
	hipError_t err = hipMalloc((void**)&data, total_elements * sizeof(float));
	if (err != hipSuccess) {
		throw runtime_error("hipMalloc failed! (code M0004)");
	}
	int threadsPerBlock = 0;
	int blocksPerGrid = 0;
	switch (type)
	{
	case Zero:
		hipMemset(data, 0, total_elements * sizeof(float));
		break;
	case Ones:
		threadsPerBlock = autoSetBlockSize(ones_matrix_kernel);
		blocksPerGrid = (total_elements + threadsPerBlock - 1) / threadsPerBlock;
		ones_matrix_kernel << <blocksPerGrid, threadsPerBlock >> > (data, total_elements);
		break;
	case Identity:
		hipMemset(data, 0, total_elements * sizeof(float));
		threadsPerBlock = autoSetBlockSize(identity_matrix_kernel);
		blocksPerGrid = (total_elements + threadsPerBlock - 1) / threadsPerBlock;
		identity_matrix_kernel << <blocksPerGrid, threadsPerBlock >> > (data, rows, cols);
		break;
	case Random:
		hiprandState* state = nullptr;
		hipError_t err = hipMalloc((void**)&state, total_elements * sizeof(hiprandState));
		if (err != hipSuccess) {
			throw runtime_error("hipMalloc failed for hiprandState! (code M0005)");
		}
		threadsPerBlock = autoSetBlockSize(random_kernel);
		blocksPerGrid = (total_elements + threadsPerBlock - 1) / threadsPerBlock;
		random_kernel << <blocksPerGrid, threadsPerBlock >> > (state, (unsigned long)time(0));
		generate_random_numbers << <blocksPerGrid, threadsPerBlock >> > (state, data, total_elements);
		hipFree(state);
		break;
	}
}

cudaMatrix::cudaMatrix(const cudaMatrix& others) : rows(others.rows), cols(others.cols) {
	hipError_t err = hipMalloc((void**)&this->data, static_cast<size_t>(rows) * cols * sizeof(float));
	if (err != hipSuccess) {
		throw runtime_error("hipMalloc failed! (code M0006)");
	}
	hipMemcpy(this->data, others.data, static_cast<size_t>(rows) * cols * sizeof(float), hipMemcpyDeviceToDevice);
}

cudaMatrix::~cudaMatrix() {
	hipMemset(data, 0, static_cast<size_t>(rows) * cols * sizeof(float));
	hipFree(data);
	rows = 0;
	cols = 0;
}

cudaMatrix cudaMatrix::fromFloat(float value) {
	cudaMatrix result(1);
	hipMemcpy(result.data, &value, sizeof(float), hipMemcpyHostToDevice);
	return result;
}

void cudaMatrix::resize(int rows, int cols) {
	if (this->rows == rows && this->cols == cols) { return; }
	float* new_data = nullptr;
	hipMalloc((void**)&new_data, static_cast<size_t>(rows) * cols * sizeof(float));
	hipMemset(new_data, 0, static_cast<size_t>(rows) * cols * sizeof(float));
	dim3 threadsPerBlock = autoSetBlockSize2D(reshape_kernel, rows, cols);
	dim3 blocksPerGrid = dim3((rows + threadsPerBlock.x - 1) / threadsPerBlock.x, (cols + threadsPerBlock.y - 1) / threadsPerBlock.y);
	reshape_kernel << <blocksPerGrid, threadsPerBlock >> > (data, new_data, this->rows, this->cols, rows, cols);
	hipFree(data);
	data = new_data;
	this->rows = rows;
	this->cols = cols;
}

cudaMatrix cudaMatrix::zeros(int rows, int cols) { return cudaMatrix(rows, cols); }

cudaMatrix cudaMatrix::zeros(int size) { return cudaMatrix(size); }

cudaMatrix cudaMatrix::ones(int rows, int cols) { return cudaMatrix(rows, cols, Ones); }

cudaMatrix cudaMatrix::ones(int size) { return cudaMatrix(size, Ones); }

cudaMatrix cudaMatrix::identity(int size) { return cudaMatrix(size, Identity); }

cudaMatrix cudaMatrix::random(int rows, int cols) { return cudaMatrix(rows, cols, Random); }

cudaMatrix cudaMatrix::random(int size) { return cudaMatrix(size, Random); }

cudaMatrix cudaMatrix::operator=(const cudaMatrix& B) {
	if (this == &B) { return *this; }
	if (rows != B.rows || cols != B.cols) {
		hipFree(data);
		hipMalloc((void**)&data, static_cast<size_t>(B.rows) * B.cols * sizeof(float));
		rows = B.rows;
		cols = B.cols;
	}
	hipMemcpy(data, B.data, static_cast<size_t>(B.rows) * B.cols * sizeof(float), hipMemcpyDeviceToDevice);
	return *this;
}

void cudaMatrix::set(int row, int col, float value) {
	if (row < 0 || row >= rows || col < 0 || col >= cols) {
		throw out_of_range("索引超出范围。");
	}
	hipMemcpy(data + static_cast<size_t>(row) * cols + col, &value, sizeof(float), hipMemcpyHostToDevice);
}

void cudaMatrix::setData(const vector<float> v) { hipMemcpy(data, v.data(), v.size() * sizeof(float), hipMemcpyHostToDevice); }

float cudaMatrix::get(int row, int col) const {
	if (row < 0 || row >= rows || col < 0 || col >= cols) {
		throw out_of_range("索引超出范围。");
	}
	float result = 0.0f;
	hipMemcpy(&result, data + static_cast<size_t>(row) * cols + col,
		sizeof(float), hipMemcpyDeviceToHost);
	return result;
}

void cudaMatrix::getData(vector<float>& v, ...) const {
	va_list args;
	va_start(args, &v);
	if (va_arg(args, bool)) { // 如果第一个参数为 true，则强制重新分配内存
		v.resize(static_cast<size_t>(rows) * cols);
	}
	va_end(args);
	hipMemcpy(v.data(), data,
		v.size() * sizeof(float), hipMemcpyDeviceToHost);
}

float* cudaMatrix::getDataPtr() const { return data; }

int cudaMatrix::getRows() const { return rows; }

int cudaMatrix::getCols() const { return cols; }

void cudaMatrix::printData() const {
	vector<float> hostData(rows * cols);
	hipMemcpy(hostData.data(), data,
		static_cast<size_t>(rows) * cols * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < rows; ++i) {
		for (int j = 0; j < cols; ++j) {
			cout << hostData[static_cast<size_t>(i) * cols + j] << " ";
		}
		cout << endl;
	}
}

float cudaMatrix::norm(int L) const {
	if (rows != 1 && cols != 1)
		throw invalid_argument("输入不是向量，无法求范数。");
	if (L <= 0)
		throw invalid_argument("范数阶数必须大于 0。");
	int size = max(rows, cols);
	//int threadsPerBlock = 768;
	int threadsPerBlock = autoSetBlockSize(norm_kernel);
	int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
	float* vec2 = nullptr;
	hipMalloc((void**)&vec2, size * sizeof(float));
	norm_kernel << <blocksPerGrid, threadsPerBlock >> > (data, vec2, size, L);
	float* result = nullptr;
	hipMalloc((void**)&result, sizeof(float));
	hipMemset(result, 0, sizeof(float));
	reduce_sum << <1, 512, 512 * sizeof(float) >> > (vec2, result, size);
	//hipDeviceSynchronize();
	float result_host = 0.0f;
	hipMemcpy(&result_host, result, sizeof(float), hipMemcpyDeviceToHost);
	hipFree(result);
	return powf(result_host, 1.0f / L);
}

cudaMatrix::operator float() const {
	if (rows != 1 || cols != 1)
		throw invalid_argument("矩阵规模不是 1x1，无法转换为 float。");
	float result = 0.0f;
	hipMemcpy(&result, data, sizeof(float), hipMemcpyDeviceToHost);
	return result;
}

bool cudaMatrix::operator<(const float n) {
	if (rows != 1 || cols != 1)
		throw invalid_argument("输入不是标量，无法比较。");
	return data[0] < n;
}

bool cudaMatrix::operator<(const cudaMatrix& B) {
	if (rows != 1 || cols != 1 || B.rows != 1 || B.cols != 1)
		throw invalid_argument("输入不是标量，无法比较。");
	return data[0] < B.data[0];
}

bool cudaMatrix::operator<=(const float n) {
	if (rows != 1 || cols != 1)
		throw invalid_argument("输入不是标量，无法比较。");
	return data[0] <= n;
}

bool cudaMatrix::operator<=(const cudaMatrix& B) {
	if (rows != 1 || cols != 1 || B.rows != 1 || B.cols != 1)
		throw invalid_argument("输入不是标量，无法比较。");
	return data[0] <= B.data[0];
}

bool cudaMatrix::operator>(const float n) {
	if (rows != 1 || cols != 1)
		throw invalid_argument("输入不是标量，无法比较。");
	return data[0] > n;
}

bool cudaMatrix::operator>(const cudaMatrix& B) {
	if (rows != 1 || cols != 1 || B.rows != 1 || B.cols != 1)
		throw invalid_argument("输入不是标量，无法比较。");
	return data[0] > B.data[0];
}

bool cudaMatrix::operator>=(const float n) {
	if (rows != 1 || cols != 1)
		throw invalid_argument("输入不是标量，无法比较。");
	return data[0] >= n;
}

bool cudaMatrix::operator>=(const cudaMatrix& B) {
	if (rows != 1 || cols != 1 || B.rows != 1 || B.cols != 1)
		throw invalid_argument("输入不是标量，无法比较。");
	return data[0] >= B.data[0];
}

void cudaMatrix::add(cudaMatrix& B) {
	if (rows != B.rows || cols != B.cols) {
		throw invalid_argument("矩阵维度不匹配，无法相加。");
	}
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	const float alpha = 1.0f;
	const float beta = 1.0f;
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		rows, cols,
		&alpha, data, rows,
		&beta, B.data, B.rows,
		data, rows);
	hipblasDestroy(handle);
}

cudaMatrix cudaMatrix::add(cudaMatrix& A, cudaMatrix& B) {
	if (A.rows != B.rows || A.cols != B.cols) {
		throw invalid_argument("矩阵维度不匹配，无法相加。");
	}
	cudaMatrix result(A.rows, A.cols);
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	const float alpha = 1.0f;
	const float beta = 1.0f;

	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		A.rows, A.cols,
		&alpha, A.data, A.rows,
		&beta, B.data, B.rows,
		result.data, result.rows);

	hipblasDestroy(handle);
	return result;
}

cudaMatrix cudaMatrix::operator+(cudaMatrix& B) { return add(*this, B); }

cudaMatrix cudaMatrix::operator+=(const cudaMatrix& B) {
	this->add(const_cast<cudaMatrix&>(B));
	return *this;
}

void cudaMatrix::subtract(cudaMatrix& B) {
	if (rows != B.rows || cols != B.cols) {
		throw invalid_argument("矩阵维度不匹配，无法相减。");
	}
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	const float alpha = 1.0f;
	const float beta = -1.0f;
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		rows, cols,
		&alpha, data, rows,
		&beta, B.data, B.rows,
		data, rows);
	hipblasDestroy(handle);
}

cudaMatrix cudaMatrix::subtract(const cudaMatrix& A, const cudaMatrix& B) {
	if (A.rows != B.rows || A.cols != B.cols) {
		throw invalid_argument("矩阵维度不匹配，无法相减。");
	}
	cudaMatrix result(A.rows, A.cols);
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	const float alpha = 1.0f;
	const float beta = -1.0f;

	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		A.rows, A.cols,
		&alpha, A.data, A.rows,
		&beta, B.data, B.rows,
		result.data, result.rows);

	hipblasDestroy(handle);
	return result;
}

cudaMatrix cudaMatrix::operator-(const cudaMatrix& B) { return subtract(*this, B); }

cudaMatrix cudaMatrix::operator-=(const cudaMatrix& B) {
	this->subtract(const_cast<cudaMatrix&>(B));
	return *this;
}

void cudaMatrix::multiply(cudaMatrix& B) {
	if (rows == 1 && cols == 1) {
		hipblasHandle_t handle;
		hipblasCreate(&handle);
		hipblasSscal(handle, rows * cols, data, B.data, 1);
		hipblasDestroy(handle);
		return;
	}
	if (cols != B.rows) {
		throw invalid_argument("矩阵维度不匹配，无法相乘。");
	}
	cudaMatrix temp(rows, B.cols);
	cudaMatrix tempA = transpose();
	cudaMatrix tempB = B.transpose();
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	const float alpha = 1.0f;
	const float beta = 0.0f;
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		rows, B.cols, cols,
		&alpha, tempA.data, tempA.cols,
		tempB.data, tempB.cols,
		&beta, temp.data, temp.rows);
	int tmp = temp.cols;
	temp.cols = temp.rows;
	temp.rows = tmp;
	hipMemcpy(data, temp.transpose().data, static_cast<size_t>(rows) * cols * sizeof(float), hipMemcpyDeviceToDevice);
	hipblasDestroy(handle);
}

cudaMatrix cudaMatrix::multiply(const cudaMatrix& A, const cudaMatrix& B) { // 请不要管这里，这里是屎山
	if (A.rows == 1 && A.cols == 1) {
		cudaMatrix result(B);
		hipblasHandle_t handle;
		hipblasCreate(&handle);
		hipblasSscal(handle, B.rows * B.cols, A.data, result.data, 1);
		hipblasDestroy(handle);
		return result;
	}
	if (A.cols != B.rows) {
		throw invalid_argument("矩阵维度不匹配，无法相乘。");
	}
	cudaMatrix temp(A.rows, B.cols);
	cudaMatrix tempA = A.transpose();
	cudaMatrix tempB = B.transpose();
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	const float alpha = 1.0f;
	const float beta = 0.0f;

	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		A.rows, B.cols, A.cols,
		&alpha, tempA.data, tempA.cols,
		tempB.data, tempB.cols,
		&beta, temp.data, temp.rows);
	int tmp = temp.cols;
	temp.cols = temp.rows;
	temp.rows = tmp;
	cudaMatrix result = temp.transpose();
	hipblasDestroy(handle);
	return result;
}

cudaMatrix operator*(const cudaMatrix& A, const cudaMatrix& B) { return cudaMatrix::multiply(A, B); }

cudaMatrix operator*(float scalar, const cudaMatrix& A) { return A.scalarMultiply(scalar); }

cudaMatrix operator*(const cudaMatrix& A, const float scalar) { return A.scalarMultiply(scalar); }

cudaMatrix cudaMatrix::operator*=(const cudaMatrix& B) {
	this->multiply(const_cast<cudaMatrix&>(B));
	return *this;
}

cudaMatrix cudaMatrix::operator*=(const float scalar) {
	this->scalarMultiply(scalar);
	return *this;
}

cudaMatrix cudaMatrix::operator^(int pows) {
	if (rows != cols) {
		throw invalid_argument("矩阵不是方阵，无法求幂。");
	}
	if (pows < 0) {
		throw invalid_argument("幂次必须大于等于 0。");
	}
	cudaMatrix result(rows, cols, Identity);
	cudaMatrix base = *this;
	while (pows > 0) {
		if (pows % 2 == 1) {
			result = multiply(result, base);
		}
		base = multiply(base, base);
		pows /= 2;
	}
	return result;
}

cudaMatrix cudaMatrix::operator^=(int pows) {
	if (rows != cols) {
		throw invalid_argument("矩阵不是方阵，无法求幂。");
	}
	if (pows < 0) {
		throw invalid_argument("幂次必须大于等于 0。");
	}
	pows--;
	cudaMatrix base = *this;
	while (pows > 0) {
		if (pows % 2 == 1) {
			*this = multiply(*this, base);
		}
		base = multiply(base, base);
		pows /= 2;
	}
	return *this;
}

cudaMatrix cudaMatrix::transpose() const {
	cudaMatrix result(cols, rows);
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	const float alpha = 1.0f;
	const float beta = 0.0f;

	hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
		rows, cols,
		&alpha, data, cols,
		&beta, data, rows,
		result.data, result.cols);

	hipblasDestroy(handle);
	return result;
}

cudaMatrix cudaMatrix::transpose(const cudaMatrix& A) { return A.transpose(); }

cudaMatrix cudaMatrix::operator~() const { return this->transpose(); }

float cudaMatrix::trace() const {
	if (rows != cols) {
		throw invalid_argument("矩阵不是方阵，无法求迹。");
	}
	int size = rows;
	float* trace_array = nullptr;
	hipMalloc((void**)&trace_array, size * sizeof(float));
	int threadsPerBlock = autoSetBlockSize(get_diag);
	int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
	get_diag << <blocksPerGrid, threadsPerBlock >> > (data, trace_array, size);
	hipDeviceSynchronize();
	float* d_result = nullptr;
	hipMalloc((void**)&d_result, sizeof(float));
	hipMemset(d_result, 0, sizeof(float));
	reduce_sum << <1, 512, 512 * sizeof(float) >> > (trace_array, d_result, size);
	float result = 0.0;
	hipMemcpy(&result, d_result, sizeof(float), hipMemcpyDeviceToHost);
	hipFree(trace_array);
	hipFree(d_result);
	return (float)result;
}

float cudaMatrix::trace(const cudaMatrix& A) { return A.trace(); }

cudaMatrix cudaMatrix::scalarMultiply(float scalar) const {
	cudaMatrix result(rows, cols);
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipMemcpy(result.data, data, static_cast<size_t>(rows) * cols * sizeof(float), hipMemcpyDeviceToDevice);
	hipblasSscal(handle, rows * cols, &scalar, result.data, 1);
	hipblasDestroy(handle);
	return result;
}

cudaMatrix cudaMatrix::matrixDOTmatrix(const cudaMatrix& A, const cudaMatrix& B) {
	if (A.rows != B.rows || A.cols != B.cols) {
		throw invalid_argument("矩阵维度不匹配，无法进行点乘。");
	}
	cudaMatrix result(A.rows, A.cols);
	int size = A.rows * A.cols;
	int threadsPerBlock = autoSetBlockSize(elementwise_multiply_kernel);
	int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
	elementwise_multiply_kernel << <blocksPerGrid, threadsPerBlock >> > (A.data, B.data, result.data, size);
	return result;
}

cudaMatrix cudaMatrix::vectorBroadcast2Matrix(const cudaMatrix& sourceVector, const int rows, const int cols) {
	if (sourceVector.cols != 1 && sourceVector.rows != 1) {
		throw invalid_argument("输入矩阵不是向量，无法进行广播。");
	}
	if (rows <= 0 || cols <= 0) {
		throw invalid_argument("广播长度必须大于 0。");
	}
	int size = rows * cols;
	cudaMatrix result(rows, cols);
	if (sourceVector.cols == 1) {
		if (sourceVector.rows != rows)
			throw invalid_argument("向量长度与广播长度不匹配。");
		int threadsPerBlock = autoSetBlockSize(col_vec_broadcast2matrix_kernel);
		int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
		col_vec_broadcast2matrix_kernel << <blocksPerGrid, threadsPerBlock >> > (sourceVector.data, result.data, cols, size);
	}
	else if (sourceVector.rows == 1) {
		if (sourceVector.cols != cols)
			throw invalid_argument("向量长度与广播长度不匹配。");
		int threadsPerBlock = autoSetBlockSize(row_vec_broadcast2matrix_kernel);
		int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
		row_vec_broadcast2matrix_kernel << <blocksPerGrid, threadsPerBlock >> > (sourceVector.data, result.data, cols, size);
	}
	return result;
}

cudaMatrix cudaMatrix::dot(const cudaMatrix& A, const cudaMatrix& B) {
	if (A.rows == B.rows && A.cols == B.cols) {
		return matrixDOTmatrix(A, B);
	}
	int row = A.rows > B.rows ? A.rows : B.rows;
	int col = A.cols > B.cols ? A.cols : B.cols;
	cudaMatrix tempA(row, col);
	cudaMatrix tempB(row, col);
	if (A.cols == 1 || A.rows == 1) {
		cudaMatrix temp1 = vectorBroadcast2Matrix(A, row, col);
		hipMemcpy(tempA.data, temp1.data, static_cast<size_t>(row) * col * sizeof(float), hipMemcpyDeviceToDevice);
	}
	else
		hipMemcpy(tempA.data, A.data, static_cast<size_t>(A.rows) * A.cols * sizeof(float), hipMemcpyDeviceToDevice);
	if (B.cols == 1 || B.rows == 1) {
		cudaMatrix temp2 = vectorBroadcast2Matrix(B, row, col);
		hipMemcpy(tempB.data, temp2.data, static_cast<size_t>(row) * col * sizeof(float), hipMemcpyDeviceToDevice);
	}
	else
		hipMemcpy(tempB.data, B.data, static_cast<size_t>(row) * col * sizeof(float), hipMemcpyDeviceToDevice);
	return matrixDOTmatrix(tempA, tempB);
}

cudaMatrix cudaMatrix::dot(const float scalar, const cudaMatrix& A) { return A.scalarMultiply(scalar); }

cudaMatrix cudaMatrix::dot(const cudaMatrix& A, const float scalar) { return A.scalarMultiply(scalar); }

cudaMatrix cudaMatrix::divide(const cudaMatrix& A, const cudaMatrix& B) {
	if (A.rows != B.rows || A.cols != B.cols) {
		throw invalid_argument("矩阵维度不匹配，无法相除。");
	}
	cudaMatrix result(A.rows, A.cols);
	int size = A.rows * A.cols;
	int threadsPerBlock = autoSetBlockSize(divide_kernel);
	int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
	divide_kernel << <blocksPerGrid, threadsPerBlock >> > (A.data, B.data, result.data, size);
	return result;
}

cudaMatrix cudaMatrix::operator/(const cudaMatrix& B) { return divide(*this, B); }

cudaMatrix cudaMatrix::operator/(const float scalar) {
	float invScalar = 1.0f / scalar;
	cudaMatrix result(rows, cols);
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipMemcpy(result.data, data, static_cast<size_t>(rows) * cols * sizeof(float), hipMemcpyDeviceToDevice);
	hipblasSscal(handle, rows * cols, &invScalar, result.data, 1);
	hipblasDestroy(handle);
	return result;
}

cudaMatrix cudaMatrix::operator/=(const cudaMatrix& B) {
	if (rows != B.rows || cols != B.cols) {
		throw invalid_argument("矩阵维度不匹配，无法相除。");
	}
	int size = rows * cols;
	int threadsPerBlock = autoSetBlockSize(divide_kernel);
	int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
	divide_kernel << <blocksPerGrid, threadsPerBlock >> > (data, B.data, data, size);
	return *this;
}

cudaMatrix cudaMatrix::operator/=(const float scalar) {
	float invScalar = 1.0f / scalar;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSscal(handle, rows * cols, &invScalar, data, 1);
	hipblasDestroy(handle);
	return *this;
}

cudaMatrix cudaMatrix::solveSparseSLE(cudaMatrix& A, cudaMatrix& b) {
	if (A.rows != b.rows) {
		throw invalid_argument("矩阵维度不匹配，无法求解稀疏线性方程组。");
	}
	if (b.cols != 1) {
		throw invalid_argument("右侧矩阵不是列向量，无法求解稀疏线性方程组。");
	}
	cudaMatrix x(A.rows, 1, Ones);
	cudaMatrix r = b - A * x;
	cudaMatrix p = r;
	cudaMatrix r_old = r;
	float r_norm2 = ~r * r;
	for (int i = 0; i < 1e6; i++)
	{
		cudaMatrix Ap = A * p;
		float alpha = r_norm2 / (~p * Ap);
		x = alpha * p + x;
		r = r - alpha * Ap;
		float beta = (~r * r) / (~r_old * r_old);
		p = beta * p + r;
		r_old = r;
		r_norm2 = ~r * r;
		if (r_norm2 < 1e-16)
			break;
	}
	return x;
}

cudaMatrix cudaMatrix::operator| (cudaMatrix& b) { return solveSparseSLE(*this, b); }

float cudaMatrix::det() const {
	if (rows != cols) {
		throw invalid_argument("矩阵不是方阵，无法求行列式。");
	}
	int size = rows;
	hipsolverHandle_t handle;
	hipsolverDnCreate(&handle);
	cudaMatrix temp(*this);
	int* Pivots = nullptr;
	int* Info = nullptr;
	hipMalloc((void**)&Pivots, size * sizeof(int));
	hipMalloc((void**)&Info, sizeof(int));
	int workspace_size = 0;
	hipsolverDnSgetrf_bufferSize(handle, size, size, temp.data, size, &workspace_size);
	float* workspace = nullptr;
	hipMalloc((void**)&workspace, workspace_size * sizeof(float));
	hipsolverDnSgetrf(handle, size, size, temp.data, size, workspace, Pivots, Info);
	float det = 1.0f;
	float* diag = nullptr;
	hipMalloc((void**)&diag, size * sizeof(float));
	int threadsPerBlock = autoSetBlockSize(get_diag);
	int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
	get_diag << <blocksPerGrid, threadsPerBlock >> > (temp.data, diag, size);
	hipDeviceSynchronize();
	vector<float> hostDiag(size);
	hipMemcpy(hostDiag.data(), diag, size * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < size; ++i) { det *= hostDiag[i]; }
	vector<int> hostPivots(size);
	hipMemcpy(hostPivots.data(), Pivots, size * sizeof(int), hipMemcpyDeviceToHost);
	int pivotSign = 1;
	for (int i = 0; i < size; ++i) {
		if (hostPivots[i] != (i + 1))
			pivotSign *= -1;
	}
	det *= pivotSign;
	hipFree(Pivots);
	hipFree(Info);
	hipFree(workspace);
	hipsolverDnDestroy(handle);
	return det;
}

float cudaMatrix::det(const cudaMatrix& A) { return A.det(); }

cudaMatrix cudaMatrix::diag(vector<int> offset, ...) {
	int num = offset.size();
	va_list args;
	va_start(args, offset);
	vector<vector<float>> arg(num);
	for (int i = 0; i < num; i++) {
		arg[i] = va_arg(args, vector<float>);
		if (arg[i].data() == nullptr)
			throw invalid_argument("输入矩阵指针为空。");
	}
	va_end(args);
	int size = arg[0].size();
	cudaMatrix result(size);
	int threadsPerBlock = autoSetBlockSize(fill_diag_kernel);
	int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
	for (int i = 0; i < num; i++) {
		float* tmp = nullptr;
		hipMalloc((void**)&tmp, arg[i].size() * sizeof(float));
		hipMemcpy(tmp, arg[i].data(), arg[i].size() * sizeof(float), hipMemcpyHostToDevice);
		fill_diag_kernel << <blocksPerGrid, threadsPerBlock >> > (result.data, tmp, offset[i], size);
	}
	return result;
}

cudaMatrix::ElementProxy cudaMatrix::operator()(int row, int col) {
	if (row < 0 || row >= rows || col < 0 || col >= cols) {
		throw std::out_of_range("索引超出范围。");
	}
	return ElementProxy(*this, row, col);
}

cudaMatrix::ElementProxy::~ElementProxy() {
	row = 0;
	col = 0;
	mat = cudaMatrix();
}

cudaMatrix::ElementProxy::operator float() const { return mat.get(row, col); }

cudaMatrix::ElementProxy& cudaMatrix::ElementProxy::operator=(float value) {
	mat.set(row, col, value);
	return *this;
}

cudaMatrix cudaMatrix::assembleBlocks(const vector<vector<cudaMatrix>>& block) {
	if (block.empty()) {
		throw std::invalid_argument("Blocks cannot be empty");
	}
	vector<vector<cudaMatrix>> blocks = block;
	int numBlockRows = blocks.size();
	int numBlockCols = blocks[0].size();
	for (int i = 0; i < numBlockRows; ++i) {
		if (blocks[i].size() != numBlockCols) {
			throw std::invalid_argument("All rows must have the same number of blocks");
		}
	}
	vector<int> maxBlockRows(numBlockRows, 0);
	vector<int> maxBlockCols(numBlockCols, 0);
	for (int i = 0; i < numBlockRows; ++i) {
		for (int j = 0; j < numBlockCols; ++j) {
			int blockRows = blocks[i][j].getRows();
			int blockCols = blocks[i][j].getCols();
			if (blockRows > maxBlockRows[i])
				maxBlockRows[i] = blockRows;
			if (blockCols > maxBlockCols[j])
				maxBlockCols[j] = blockCols;
		}
	}
	for (int i = 0; i < numBlockRows; ++i) {
		for (int j = 0; j < numBlockCols; ++j) {
			blocks[i][j].resize(maxBlockRows[i], maxBlockCols[j]);
		}
	}
	int totalRows = 0;
	for (int i = 0; i < numBlockRows; ++i) {
		totalRows += maxBlockRows[i];
	}
	int totalCols = 0;
	for (int j = 0; j < numBlockCols; ++j)
		totalCols += maxBlockCols[j];
	cudaMatrix result(totalRows, totalCols);
	int rowOffset = 0;
	for (int i = 0; i < numBlockRows; ++i) {
		int colOffset = 0;
		for (int j = 0; j < numBlockCols; ++j) {
			const cudaMatrix& block = blocks[i][j];
			int blockRows = block.getRows();
			int blockCols = block.getCols();
			const float* srcData = block.getDataPtr();
			float* destData = result.getDataPtr() + (rowOffset * totalCols + colOffset);
			size_t srcPitch = blockCols * sizeof(float);
			size_t destPitch = totalCols * sizeof(float);
			size_t widthInBytes = blockCols * sizeof(float);
			size_t height = blockRows;
			hipError_t err = hipMemcpy2D(destData, destPitch,
				srcData, srcPitch,
				widthInBytes, height,
				hipMemcpyDeviceToDevice);
			if (err != hipSuccess) {
				throw runtime_error("hipMemcpy2D failed: " + string(hipGetErrorString(err)));
			}
			colOffset += blockCols;
		}
		rowOffset += maxBlockRows[i];
	}
	return result;
}

cudaMatrix cudaMatrix::setdiff(const cudaMatrix& A, const cudaMatrix& B) {
	int sizeA = A.rows * A.cols;
	int sizeB = B.rows * B.cols;
	cudaMatrix result(1, sizeA);

	int threadsPerBlock = autoSetBlockSize(setdiff_kernel);
	int blocksPerGrid = (sizeA + threadsPerBlock - 1) / threadsPerBlock;

	setdiff_kernel << <blocksPerGrid, threadsPerBlock >> > (A.data, B.data, result.data, sizeA, sizeB);

	// Remove NaN values from the result
	vector<float> hostResult(sizeA);
	hipMemcpy(hostResult.data(), result.data, sizeA * sizeof(float), hipMemcpyDeviceToHost);
	hostResult.erase(remove_if(hostResult.begin(), hostResult.end(), [](float val) { return isnan(val); }), hostResult.end());

	cudaMatrix finalResult(1, hostResult.size());
	hipMemcpy(finalResult.data, hostResult.data(), hostResult.size() * sizeof(float), hipMemcpyHostToDevice);

	return finalResult;
}

cudaMatrix cudaMatrix::subMatrix(const vector<int>& row_indices, const vector<int>& col_indices) const {
	int sub_rows = row_indices.size();
	int sub_cols = col_indices.size();

	// 创建结果矩阵
	cudaMatrix result(sub_rows, sub_cols);

	// 分配并拷贝行索引到设备端
	int* d_row_indices = nullptr;
	hipMalloc(&d_row_indices, sub_rows * sizeof(int));
	hipMemcpy(d_row_indices, row_indices.data(), sub_rows * sizeof(int), hipMemcpyHostToDevice);

	// 分配并拷贝列索引到设备端
	int* d_col_indices = nullptr;
	hipMalloc(&d_col_indices, sub_cols * sizeof(int));
	hipMemcpy(d_col_indices, col_indices.data(), sub_cols * sizeof(int), hipMemcpyHostToDevice);

	// 设置 CUDA 内核参数
	int total_elements = sub_rows * sub_cols;
	int threadsPerBlock = autoSetBlockSize(extractSubMatrixKernel);
	int blocksPerGrid = (total_elements + threadsPerBlock - 1) / threadsPerBlock;
	extractSubMatrixIndexedKernel << <blocksPerGrid, threadsPerBlock >> >
		(this->data, result.data, d_row_indices, d_col_indices, this->cols, sub_rows, sub_cols);

	// 释放设备内存
	hipFree(d_row_indices);
	hipFree(d_col_indices);

	// 返回结果矩阵
	return result;
}
